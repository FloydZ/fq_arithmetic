
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <stdio.h>
#include <string.h>
#include <stdint.h>

// TODO not really corrext
__constant__ uint64_t __p64[4] = { 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFF7BC6BFFF, 0x8AF43417A1087212, 0x01248A1B00000000 };
__constant__ uint64_t __mu64[4] = { 0x0000000100000000, 0x000000007BC6C000, 0x8AF43417A1087212, 0x91248A1B00000000 };

// just testing
uint64_t __cpu_p64[4] = { 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFF7BC6BFFF, 0x8AF43417A1087212, 0x01248A1B00000000 };
uint64_t __cpu_mu64[4] = { 0x0000000100000000, 0x000000007BC6C000, 0x8AF43417A1087212, 0x91248A1B00000000 };


// jeah i mean not really a karatsuba multiplication, but never the less a usefull
// function. c0c1 = a0*b0
#define __karatsuba64_1x1(c0,c1,a0,b0)		\
	asm volatile (  "{\n\t"					\
			"mul.lo.u64 %0, %2, %3;\n\t"	\
			"mul.hi.u64 %1, %2, %3;\n\t"	\
			"}\n\t" 						\
		: "=l"(c0), "=l"(c1)				\
		: "l"(a0), "l"(b0));					

// 2limbs times 2limbs karatsuba multiplication 
#define __karatsuba64_2x2(c0,c1,c2,c3,a0,a1,b0,b1) 					\
	asm volatile("{\n\t"                                            \
			".reg .u64 Al, Bl, Ah, Bh, t0h, t1l, t1m, t2l;\n\t" 	\
			".reg .u64 t1m1, t1m2;\n\t" 	 						\
			"add.cc.u64	   	   Al, %4, %5 ;\n\t"					\
			"addc.u64          Ah,  0,  0 ;\n\t"					\
			"add.cc.u64	       Bl, %6, %7 ;\n\t"					\
			"addc.u64          Bh,  0,  0 ;\n\t"					\
			"mul.lo.u64        t1l, Al, Bl;\n\t" 					\
			"mul.hi.u64        t1m, Al, Bl;\n\t" 					\
			"mul.lo.u64        t1m1, Ah, Bl;\n\t"	 				\
			"mul.lo.u64        t1m2, Al, Bh;\n\t"					\
			"add.u64           t1m, t1m, t1m1;\n\t" 				\
			"add.u64           t1m, t1m, t1m2;\n\t" 				\
			"mul.lo.u64        %0,  %4, %6;\n\t"                    \
			"mul.hi.u64        t0h, %4, %6;\n\t" 					\
			"mul.lo.u64        t2l, %5, %7;\n\t" 					\
			"mul.hi.u64        %3,  %5, %7;\n\t" 					\
			/*AB - a1b1*/ 											\
			"sub.cc.u64        t1l, t1l, t2l;\n\t"                  \
			"subc.u64          t1m, t1m, %3 ;\n\t"                  \
			/*AB - a1b1 - a0b0*/ 									\
			"sub.cc.u64        t1l, t1l, %0 ;\n\t"                  \
			"subc.u64          t1m, t1m, t0h;\n\t"                  \
			"add.cc.u64 	   %1, t1l, t0h ;\n\t"					\
			"addc.u64 	   %2, t1m, t2l ;\n\t"						\
			"}\n"                                                   \
		: "=l"(c0), "=l"(c1), "=l"(c2), "=l"(c3) 	/*0,3*/     	\
		: "l"(a0), "l"(a1),    			 			/*4,5*/     	\
		  "l"(b0), "l"(b1),  			 			/*6,7*/     	\
		  "l"(c0), "l"(c1), "l"(c2), "l"(c3));   	/*8,11*/

// 1limb times 2limbs karatsuba multiplication 
#define __karatsuba64_1x2(c0,c1,c2,a0,b0,b1)\
	asm volatile("{\n\t"                                    \
			".reg .u64 Bl, Bh, t0h, t1l, t1m;\n\t" 			\
			"add.cc.u64	       Bl, %4, %5 ;\n\t"			\
			"addc.u64          Bh,  0,  0 ;\n\t"			\
			"mul.lo.u64        t1l, %3, Bl;\n\t" 			\
			"mul.hi.u64        t1m, %3, Bl;\n\t" 			\
			"mad.lo.cc.u64     t1m, %3, Bh, t1m;\n\t"		\
			"madc.hi.u64       %2,  %3, Bh, 0;\n\t" 		\
			"mul.lo.u64        %0,  %3, %4;\n\t"            \
			"mul.hi.u64        t0h, %3, %4;\n\t" 			\
			/*AB - a0b0*/ 									\
			"sub.cc.u64        t1l, t1l, %0 ;\n\t"          \
			"subc.cc.u64       t1m, t1m, t0h;\n\t"          \
			"subc.u64          %2, %8, 0    ;\n\t"          \
			"add.cc.u64 	   %1, t1l, t0h ;\n\t"			\
			"addc.u64 	       %2, t1m, 0   ;\n\t"			\
			"}\n"                                           \
		: "=l"(c0), "=l"(c1), "=l"(c2) 	/*0,2*/         	\
		: "l"(a0),    			 		/*3  */         	\
		  "l"(b0), "l"(b1) 			 	/*4,5*/ 			\
		  "l"(c0), "l"(c1), "l"(c2));   /*6,8*/  

// 1limb times 3limbs karatsuba multiplication 
#define __karatsuba64_1x3(c0,c1,c2,c3,a0,b0,b1,b2) 	\
	__karatsuba64_1x2(c0,c1,c2,a0,b0,b1) 			\
	asm volatile("{\n\t" 							\
			".reg .u64 t0, t1;\n\t" 				\
			"mul.lo.u64 t0, %2, %3;\n\t" 			\
			"mul.hi.u64 t1, %2, %3;\n\t" 			\
			"add.cc.u64 %0, %4, t0;\n\t" 			\
			"addc.u64   %1, %5, t1;\n\t" 			\
			"}\n" 									\
		: "=l"(c2), "=l"(c3) /*0,1*/ 				\
		: "l"(a0), "l"(b2),  /*2,3*/ 				\
		  "l"(c2), "l"(c3)); /*3,5*/

// 1limb times 4limbs karatsuba multiplication 
#define __karatsuba64_1x4(c0,c1,c2,c3,c4,a0,b0,b1,b2,b3) 	\
	__karatsuba64_1x3(c0,c1,c2,c3,a0,b0,b1,b2) 				\
	asm volatile("{\n\t" 									\
			".reg .u64 t0, t1;\n\t" 						\
			"mul.lo.u64 t0, %2, %3;\n\t" 					\
			"mul.hi.u64 t1, %2, %3;\n\t" 					\
			"add.cc.u64 %0, %4, t0;\n\t" 					\
			"addc.u64   %1, %5, t1;\n\t" 					\
			"}\n" 											\
		: "=l"(c3), "=l"(c4) /*0,1*/ 						\
		: "l"(a0), "l"(b3),  /*2,3*/ 						\
		  "l"(c3), "l"(c4)); /*4,5*/


// adds c0c1 = a0+b0b1
// needed for the karatsuba multiplication 3x3
#define __karatsuba64_3x3_2add_helper(c0,c1,a0,b0,b1) 	\
	asm volatile (	"{\n\t" 							\
			"add.cc.u64 	%0, %2, %3;\n\t"			\
			"addc.u64       %1, %4, 0 ;\n\t"			\
			"}\n\t" 									\
		: "=l"(c0), "=l"(c1) 			/*0,1*/			\
		: "l"(a0), "l"(b0), "l"(b1));   /*2,3,4*/

// karatsuba 3x3 multiplication without reduction
#define __karatsuba64_helper_3x3(c0,c1,c2,c3,c4,c5,a0,a1,a2,b0,b1,b2,kt0,kt1,st0,st1,rt0,rt1,rt2,rt3) \
	__karatsuba64_1x1(c0,c1,a0,b0)											\
	__karatsuba64_2x2(c2,c3,c4,c5,a1,a2,b1,b2) 								\
	__karatsuba64_3x3_2add_helper(kt0,kt1,a0,a1,a2)							\
	__karatsuba64_3x3_2add_helper(st0,st1,b0,b1,b2)							\
	__karatsuba64_2x2(rt0,rt1,rt2,rt3,kt0,kt1,st0,st1)						\
	/*UsVs -= U1V1*/ 														\
	asm volatile (	"{\n\t" 												\
			"sub.cc.u64  %0, %8,  %4;\n\t"  								\
			"subc.cc.u64 %1, %9,  %5;\n\t"									\
			"subc.cc.u64 %2, %10, %6;\n\t"									\
			"subc.u64    %3, %11, %7;\n\t"									\
			"}\n\t"															\
		: "=l"(rt0),"=l"(rt1),"=l"(rt2),"=l"(rt3) 	/*output 0,3 */     	\
		: "l"(c2),"l"(c3),"l"(c4),"l"(c5),        	/*U1V1   4,7 */ 		\
		  "l"(rt0),"l"(rt1),"l"(rt2),"l"(rt3)     	/*output 8,11*/   		\
	);																		\
	/*UsVs -= U0V0*/ 														\
	asm volatile (	"{\n\t" 												\
			"sub.cc.u64  %0, %6,  %4;\n\t"									\
			"subc.cc.u64 %1, %7,  %5;\n\t"									\
			"subc.cc.u64 %2, %8,   0;\n\t"									\
			"subc.u64    %3, %9,   0;\n\t"									\
			"}\n\t"															\
		: "=l"(rt0),"=l"(rt1),"=l"(rt2),"=l"(rt3) 	/*output 0,3*/   	  	\
		: "l"(c0),"l"(c1),                 	  		/*U0V0   4,5*/   	 	\
		  "l"(rt0),"l"(rt1),"l"(rt2),"l"(rt3)     	/*output 6,9*/   		\
	);																		\
	/*Final addition and result is written */ 								\
	asm volatile("{\n\t" 													\
			"add.cc.u64  %0, %8,  %4;\n\t" 									\
			"addc.cc.u64 %1, %9,  %5;\n\t" 									\
			"addc.cc.u64 %2, %10, %6;\n\t" 									\
			"addc.u64    %3, %11, %7;\n\t" 									\
			"}\n" 															\
		: "=l"(c1),"=l"(c2),"=l"(c3),"=l"(c4) /*0, 3*/    					\
		: "l"(rt0),"l"(rt1),"l"(rt2),"l"(rt3),/*4, 7*/						\
		  "l"(c1),"l"(c2),"l"(c3),"l"(c4));   /*8,11*/

// TODO explain
#define __karatsuba64_3x3(c0,c1,c2,c3,c4,c5,a0,a1,a2,b0,b1,b2) 	\
	uint64_t kt0,kt1,st0,st1,rt0,rt1,rt2,rt3;					\
	__karatsuba64_helper_3x3(c0,c1,c2,c3,c4,c5,a0,a1,a2,b0,b1,b2,kt0,kt1,st0,st1,rt0,rt1,rt2,rt3)

					
// c = a+b, where a and b are only two limb wide
#define __karatsuba64_4x4_2add_helper(c0,c1,c2,a0,a1,b0,b1) 	\
	asm volatile("{\n\t" 										\
			"add.cc.u64  %0, %3, %5;\n\t" 						\
			"addc.cc.u64 %1, %4, %6;\n\t" 						\
			"addc.u64    %2,  0,  0;\n\t" 						\
			"}\n" 												\
		: "=l"(c0), "=l"(c1), "=l"(c2)         /*0,2*/			\
		: "l"(a0), "l"(a1), "l"(b0), "l"(b1)); /*3,6*/

// karatsuba 4x4 multiplication without reduction
#define __karatsuba64_helper_4x4(c0,c1,c2,c3,c4,c5,c6,c7,a0,a1,a2,a3,b0,b1,b2,b3,ktmp0,ktmp1,ktmp2,stmp0,stmp1,stmp2,rtmp0,rtmp1,rtmp2,rtmp3) 	\
	__karatsuba64_2x2(c0,c1,c2,c3,a0,a1,b0,b1) 									\
	__karatsuba64_2x2(c4,c5,c6,c7,a2,a3,b2,b3) 									\
	__karatsuba64_4x4_2add_helper(ktmp0,ktmp1,ktmp2,a0,a1,a2,a3) 				\
	__karatsuba64_4x4_2add_helper(stmp0,stmp1,stmp2,b0,b1,b2,b3) 				\
	__karatsuba64_2x2(rtmp0,rtmp1,rtmp2,rtmp3,ktmp0,ktmp1,stmp0,stmp1)			\
	asm volatile("{\n\t"  														\
			/*UsVs - U1V1 - U0V0*/ 												\
			"sub.cc.u64  %0, %12, %4;\n\t" 										\
			"subc.cc.u64 %1, %13, %5;\n\t" 										\
			"subc.cc.u64 %2, %14, %6;\n\t" 										\
			"subc.u64    %3, %15, %7;\n\t" 										\
			"sub.cc.u64  %0, %0,  %8;\n\t" 										\
			"subc.cc.u64 %1, %1,  %9;\n\t" 										\
			"subc.cc.u64 %2, %2, %10;\n\t" 										\
			"subc.u64    %3, %3, %11;\n\t"	 									\
		     "}\n" 																\
		: "=l"(rtmp0), "=l"(rtmp1), "=l"(rtmp2), "=l"(rtmp3) /*output 0,3*/     \
		: "l"(c4), "l"(c5), "l"(c6), "l"(c7),                /*U1V1   4,7*/ 	\
		  "l"(c0), "l"(c1), "l"(c2), "l"(c3)                 /*U0V0   8,11*/    \
		  "l"(rtmp0), "l"(rtmp1), "l"(rtmp2), "l"(rtmp3)     /*output 12,15*/   \
		); 																		\
	/*Final addition and result is written */ 									\
	asm volatile("{\n\t" 														\
			"add.cc.u64  %0, %10, %6;\n\t" 										\
			"addc.cc.u64 %1, %11, %7;\n\t" 										\
			"addc.cc.u64 %2, %12, %8;\n\t" 										\
			"addc.cc.u64 %3, %13, %9;\n\t" 										\
			"addc.cc.u64 %4, %14, 0 ;\n\t" 										\
			"addc.u64    %5, %15, 0 ;\n\t" 										\
		     "}\n" 																\
	: "=l"(c2), "=l"(c3), "=l"(c4), "=l"(c5), "=l"(c6), "=l"(c7) 	/*0,5*/ \
	: "l"(rtmp0), "l"(rtmp1), "l"(rtmp2), "l"(rtmp3) 	     		/*6,9*/	\
	  "l"(c2), "l"(c3), "l"(c4), "l"(c5), "l"(c6), "l"(c7));     	/*10,15*/


// wrapper function for the karatsube 4x4 64bit limbs multiplication, allocating the needed helper registers
#define __karatsuba64_4x4(c0,c1,c2,c3,c4,c5,c6,c7,a0,a1,a2,a3,b0,b1,b2,b3) 	\
	uint64_t ktmp0,ktmp1,ktmp2,stmp0,stmp1,stmp2,rtmp0,rtmp1,rtmp2,rtmp3; 	\
	__karatsuba64_helper_4x4(c0,c1,c2,c3,c4,c5,c6,c7,a0,a1,a2,a3,b0,b1,b2,b3,ktmp0,ktmp1,ktmp2,stmp0,stmp1,stmp2,rtmp0,rtmp1,rtmp2,rtmp3)

			
