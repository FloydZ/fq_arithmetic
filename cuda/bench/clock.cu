#include "hip/hip_runtime.h"
// run with
//  nvcc -O3 clock.cu -o clock && optirun ./clock
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <iostream>
#include <math.h>
#include "../src/repeat.h"

#define NOMAIN

#include "../src/arith.cuh"

__constant__ const uint32_t __p32[8] = { 0xFFFFFFFD, 0xAC0E7A06, 0x00000012, 0xffffffff, 0xFFFFFFFD, 0xAC0E7A06, 0x00000012, 0xffffffff };
__constant__ const uint32_t __mu32[8] = { 0x1, 0xeeeeee, 0xadddddd, 0x121212, 0x1, 0xeeeeee, 0xadddddd, 0x121212 };

// NOT CORRECT
__constant__ uint64_t __p64[4] = { 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFF7BC6BFFF, 0x8AF43417A1087212, 0x01248A1B };
__constant__ uint64_t __mu64[4] = { 0x0000000000000001, 0x000000007BC6C000, 0x8AF43417A1087212, 0x91248A1B };

__device__ __forceinline__ void fp_add96_32(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_add96_32(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2]);
}
__device__ __forceinline__ void fp_add128_32(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_add128_32(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3], __p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_add160_32(uint32_t out[5], const uint32_t arg1[5], const uint32_t arg2[5]) {
    __fp_add160_32(out[0], out[1], out[2], out[3], out[4], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4]);
}
__device__ __forceinline__ void fp_add192_32(uint32_t out[6], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __fp_add192_32(out[0], out[1], out[2], out[3], out[4], out[5], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5]);
}
__device__ __forceinline__ void fp_add224_32(uint32_t out[7], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __fp_add224_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5], __p32[6]);
}
__device__ __forceinline__ void fp_add256_32(uint32_t out[8], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __fp_add256_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], arg2[7], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5], __p32[6], __p32[7]);
}

__device__ __forceinline__ void fp_sub96_32(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_sub96_32(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2]);
}
__device__ __forceinline__ void fp_sub128_32(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_sub128_32(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3], __p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_sub160_32(uint32_t out[5], const uint32_t arg1[5], const uint32_t arg2[5]) {
    __fp_sub160_32(out[0], out[1], out[2], out[3], out[4], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4]);
}
__device__ __forceinline__ void fp_sub192_32(uint32_t out[6], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __fp_sub192_32(out[0], out[1], out[2], out[3], out[4], out[5], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5]);
}
__device__ __forceinline__ void fp_sub224_32(uint32_t out[7], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __fp_sub224_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5], __p32[6]);
}
__device__ __forceinline__ void fp_sub256_32(uint32_t out[8], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __fp_sub256_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], arg2[7], __p32[0], __p32[1], __p32[2], __p32[3], __p32[4], __p32[5], __p32[6], __p32[7]);
}


__device__ __forceinline__ void fp_add192_64(uint64_t out[3], const uint64_t arg1[3], const uint64_t arg2[3]) {
    __fp_add192_64(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p64[0], __p64[1], __p64[2]);
}
__device__ __forceinline__ void fp_sub192_64(uint64_t out[3], const uint64_t arg1[3], const uint64_t arg2[3]) {
    __fp_sub192_64(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p64[0], __p64[1], __p64[2]);
}
__device__ __forceinline__ void fp_add256_64(uint64_t out[4], const uint64_t arg1[4], const uint64_t arg2[4]) {
    __fp_add256_64(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg1[2], arg1[3],arg2[0], arg2[1], arg2[2], arg2[3], __p64[0], __p64[1], __p64[2], __p64[3]);
}
__device__ __forceinline__ void fp_sub256_64(uint64_t out[4], const uint64_t arg1[4], const uint64_t arg2[4]) {
    __fp_sub256_64(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3], __p64[0], __p64[1], __p64[2], __p64[3]);
}

__device__ __forceinline__ void fp_karatsuba1x1_32(uint32_t out[2], const uint32_t arg1[1], const uint32_t arg2[1]) {
    __karatsuba32_1x1(out[0], out[1], arg1[0], arg2[0]);
}
__device__ __forceinline__ void fp_karatsuba1x2_32(uint32_t out[3], const uint32_t arg1[1], const uint32_t arg2[2]) {
    __karatsuba32_1x2(out[0], out[1], out[2], arg1[0], arg2[0], arg2[1]);
}
__device__ __forceinline__ void fp_karatsuba1x2_32_v2(uint32_t out[3], const uint32_t arg1[1], const uint32_t arg2[2]) {
    __karatsuba32_1x2_v2(out[0], out[1], out[2], arg1[0], arg2[0], arg2[1]);
}
__device__ __forceinline__ void fp_karatsuba1x3_32(uint32_t out[4], const uint32_t arg1[1], const uint32_t arg2[3]) {
    __karatsuba32_1x3(out[0], out[1], out[2], out[3], arg1[0], arg2[0], arg2[1], arg2[2]);
}
__device__ __forceinline__ void fp_karatsuba1x4_32(uint32_t out[5], const uint32_t arg1[1], const uint32_t arg2[4]) {
    __karatsuba32_1x4(out[0], out[1], out[2], out[3], out[4], arg1[0], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void fp_karatsuba1x5_32(uint32_t out[6], const uint32_t arg1[1], const uint32_t arg2[5]) {
    __karatsuba32_1x5(out[0], out[1], out[2], out[3], out[4], out[5], arg1[0], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4]);
}
__device__ __forceinline__ void fp_karatsuba1x6_32(uint32_t out[7], const uint32_t arg1[1], const uint32_t arg2[6]) {
    __karatsuba32_1x6(out[0], out[1], out[2], out[3], out[4], out[5], out[6], arg1[0], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5]);
}
__device__ __forceinline__ void fp_karatsuba1x7_32(uint32_t out[8], const uint32_t arg1[1], const uint32_t arg2[7]) {
    __karatsuba32_1x7(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], arg1[0], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6]);
}


__device__ __forceinline__ void fp_karatsuba2x2_64(uint64_t out[2], const uint64_t arg1[2], const uint64_t arg2[2]) {
    __fp_mul64_2x2(out[0], out[1], arg1[0], arg1[1], arg2[0], arg2[1], __p64[0], __p64[1], __mu64[0], __mu64[1]);
}
__device__ __forceinline__ void fp_karatsuba3x3_64(uint64_t out[3], const uint64_t arg1[3], const uint64_t arg2[3]) {
    __fp_mul3x3_64(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p64[0], __p64[1], __p64[2], __mu64[0], __mu64[1], __mu64[2]);
}
__device__ __forceinline__ void fp_karatsuba3x3_32(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_school3x3_32(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_school3x3_32_v2(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32_v2(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_school3x3_32_v3(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32_v4(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_school3x3_32_v4(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32_v4(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_school3x3_32_v5(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __fp_mul3x3_32_v5(out[0], out[1], out[2], arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2], __p32[0], __p32[1], __p32[2], __mu32[0], __mu32[1], __mu32[2]);
}
__device__ __forceinline__ void fp_karatsuba4x4_64(uint64_t out[4], const uint64_t arg1[4], const uint64_t arg2[4]) {
    __fp_mul4x4_64(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu64[0], __mu64[1], __mu64[2], __mu64[3], __p64[0], __p64[1], __p64[2], __p64[3]);
}
__device__ __forceinline__ void fp_karatsuba4x4_32(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], __p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school4x4_32(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], 
			__p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school4x4_32_v2(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32_v2(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], 
			__p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school4x4_32_v3(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32_v3(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], 
			__p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school4x4_32_v4(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32_v4(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], 
			__p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school4x4_32_v7(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_mul4x4_32_v7(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], 
			__p32[0], __p32[1], __p32[2], __p32[3]);
}
__device__ __forceinline__ void fp_school5x5_32(uint32_t out[10], const uint32_t arg1[5], const uint32_t arg2[5]) {
    __fp_mul5x5_32(out[0], out[1], out[2], out[3], out[4],
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4],
			arg2[0], arg2[1], arg2[3], arg2[3], arg2[4],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4]);
}
__device__ __forceinline__ void fp_karatsuba6x6_32(uint32_t out[12], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __fp_mul6x6_32(out[0], out[1], out[2], out[3], out[4], out[5], 
		arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5],  arg2[0], 
		arg2[1], arg2[3], arg2[3], arg2[4], arg2[5],
		__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],
		__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5]);
}
__device__ __forceinline__ void fp_school6x6_32(uint32_t out[12], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __fp_mul6x6_32_v2(out[0], out[1], out[2], out[3], out[4], out[5], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5],  
			arg2[0], arg2[1], arg2[3], arg2[3], arg2[4], arg2[5],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5]);
}
__device__ __forceinline__ void fp_karatsuba7x7_32(uint32_t out[17], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __fp_mul7x7_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], 
			arg2[0], arg2[1], arg2[3], arg2[3], arg2[4], arg2[5], arg2[6],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],__mu32[6],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5],__p32[6]);
}
__device__ __forceinline__ void fp_school7x7_32(uint32_t out[14], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __fp_mul7x7_32_v2(out[0], out[1], out[2], out[3], out[4], out[5], out[6], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6],
			arg2[0], arg2[1], arg2[3], arg2[3], arg2[4], arg2[5], arg2[6],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],__mu32[6],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5],__p32[6]);
}
__device__ __forceinline__ void fp_karatsuba8x8_32(uint32_t out[16], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __fp_mul8x8_32(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg2[6], arg2[7],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],__mu32[6],__mu32[7],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5],__p32[6],__p32[7]);
}
__device__ __forceinline__ void fp_school8x8_32(uint32_t out[16], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __fp_mul8x8_32_v2(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg2[6], arg2[7],
			__mu32[0],__mu32[1],__mu32[2],__mu32[3],__mu32[4],__mu32[5],__mu32[6],__mu32[7],
			__p32[0],__p32[1],__p32[2],__p32[3],__p32[4],__p32[5],__p32[6],__p32[7]);
}

__device__ __forceinline__ void fp_sqr4x4_32(uint32_t out[4], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __fp_sqr4x4_32(out[0], out[1], out[2], out[3], 
			arg1[0], arg1[1], arg1[2], arg1[3],
			__mu32[0], __mu32[1], __mu32[2], __mu32[3], __p32[0], __p32[1], __p32[2], __p32[3]);
}

__device__ __forceinline__ void karatsuba_mul2x2_32(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __karatsuba32_2x2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void karatsuba_mul2x2_32_v2(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __karatsuba32_2x2_v2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_32(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __school32_2x2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_32_v2(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __school32_2x2_v2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_32_v3(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __school32_2x2_v3(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_32_v6(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __school32_2x2_v6(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_32_v7(uint32_t out[4], const uint32_t arg1[2], const uint32_t arg2[2]) {
    __school32_2x2_v7(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1], out[0], out[1], out[2], out[3]);
}
__device__ __forceinline__ void karatsuba_mul3x3_32(uint32_t out[6], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __karatsuba32_3x3(out[0], out[1], out[2], out[3], out[4], out[5], 
			arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2]);
}
__device__ __forceinline__ void school_mul3x3_32(uint32_t out[6], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __school32_3x3(out[0], out[1], out[2], out[3], out[4], out[5],
			arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2]);
}
__device__ __forceinline__ void school_mul3x3_32_v2(uint32_t out[6], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __school32_3x3_v2(out[0], out[1], out[2], out[3], out[4], out[5],
			arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2]);
}
__device__ __forceinline__ void karatsuba_mul4x4_32(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __karatsuba32_4x4(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void school_mul4x4_32(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __school32_4x4(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void school_mul4x4_32_v2(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __school32_4x4_v2(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void school_mul4x4_32_v3(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __school32_4x4_v3(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void school_mul4x4_32_v7(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __school32_4x4_v7(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[0], arg2[1], arg2[2], arg2[3]);
}
__device__ __forceinline__ void school_mul5x5_32(uint32_t out[10], const uint32_t arg1[5], const uint32_t arg2[5]) {
    __school32_5x5(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9],
			arg1[0], arg1[1], arg1[2], arg1[3], arg2[4], arg2[0], arg2[1], arg2[2], arg2[3], arg2[4]);
}
__device__ __forceinline__ void karatsuba_mul6x6_32(uint32_t out[12], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __karatsuba32_6x6(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5]);
}
__device__ __forceinline__ void school_mul6x6_32(uint32_t out[12], const uint32_t arg1[6], const uint32_t arg2[6]) {
    __school32_6x6(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11],
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5]);
}
__device__ __forceinline__ void karatsuba_mul7x7_32(uint32_t out[14], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __karatsuba32_7x7(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11], out[12], out[13],
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6]);
}
__device__ __forceinline__ void school_mul7x7_32(uint32_t out[14], const uint32_t arg1[7], const uint32_t arg2[7]) {
    __school32_7x7(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11], out[12], out[13], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6]);
}
__device__ __forceinline__ void karatsuba_mul8x8_32(uint32_t out[16], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __karatsuba32_8x8(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11], out[12], out[13], out[14], out[15], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], arg2[7]);
}
__device__ __forceinline__ void school_mul8x8_32(uint32_t out[16], const uint32_t arg1[8], const uint32_t arg2[8]) {
    __school32_8x8(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], out[9], out[10], out[11], out[12], out[13], out[14], out[15], 
			arg1[0], arg1[1], arg1[2], arg1[3], arg1[4], arg1[5], arg1[6], arg1[7], 
			arg2[0], arg2[1], arg2[2], arg2[3], arg2[4], arg2[5], arg2[6], arg2[7]);
}

__device__ __forceinline__ void school_sqr4x4_32(uint32_t out[8], const uint32_t arg1[4], const uint32_t arg2[4]) {
    __school_sqr32_4x4(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7],
			arg1[0], arg1[1], arg1[2], arg1[3]);
}


// low
__device__ __forceinline__ void school_mul3x3_low_32(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __school32_3x3_low(out[0], out[1], out[2],
			arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2]);
}
__device__ __forceinline__ void school_mul3x3_low_32_v2(uint32_t out[3], const uint32_t arg1[3], const uint32_t arg2[3]) {
    __school32_3x3_low_v2(out[0], out[1], out[2],
			arg1[0], arg1[1], arg1[2], arg2[0], arg2[1], arg2[2]);
}


// 64BIT
__device__ __forceinline__ void school_mul1x1_64_v2(uint64_t out[2], const uint64_t arg1[1], const uint64_t arg2[1]) {
    __school64_1x1_v2(out[0], out[1], arg1[0], arg2[0]);
}
__device__ __forceinline__ void school_mul1x1_64_v3(uint64_t out[2], const uint64_t arg1[1], const uint64_t arg2[1]) {
    __school64_1x1_v3(out[0], out[1], arg1[0], arg2[0]);
}

__device__ __forceinline__ void school_mul2x2_64(uint64_t out[4], const uint64_t arg1[2], const uint64_t arg2[2]) {
    __school64_2x2_v2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}
__device__ __forceinline__ void school_mul2x2_64_v2(uint64_t out[4], const uint64_t arg1[2], const uint64_t arg2[2]) {
    __school64_2x2_v2(out[0], out[1], out[2], out[3], arg1[0], arg1[1], arg2[0], arg2[1]);
}




__global__ void kclock(uint32_t *ts) {
	uint64_t start_time = 0, stop_time = 0;

	start_time = clock();
	// Measure something here
	stop_time = clock();
	
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time;
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time;
}


#define ATOMICADD32(a, b) 	atomicAdd((uint32_t *)&a, b)
#define ADD32(a, b) 		asm volatile("{ add.u32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define ADDC32(a, b) 		asm volatile("{ addc.u32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define ADDCCC32(a, b) 		asm volatile("{ addc.cc.u32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define SUB32(a, b)    	 	asm volatile("{ sub.u32 %0, %1, %2; }" : "=r"(b) : "r"(b), "r"(a))		
#define MUL32(a, b)    	 	asm volatile("{ mul.lo.u32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define MULWIDE(a,b,c) 		asm volatile("{ mul.wide.u32 %0, %1, %2; }" : "=l"(a) : "r"(b), "r"(c))		
#define MAD32(a, b)    	 	asm volatile("{ mad.lo.u32 %0, %1, %2, %0; }" : "=r"(a) : "r"(b), "r"(a))		
#define MADCCC32(a, b) 		asm volatile("{ madc.lo.cc.u32 %0, %1, %2, %0; }" : "=r"(a) : "r"(b), "r"(a))		
#define MULADDCCC32(a, b)	asm volatile("{ mul.lo.u32 %0, %1, %2; addc.cc.u32 %0, %0, %0; }" : "=r"(a) : "r"(b), "r"(a))

// src: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#data-movement-and-conversion-instructions-ld
// b must be a valid mem addr
#define LOADCONST32(a, b)   asm volatile("{ ld.const.u32  %0, [%1]; }" : "=r"(a) : "l"(b))
#define LOADGLOBAL32(a, b)  asm volatile("{ ld.global.u32 %0, [%1]; }" : "=r"(a) : "l"(b))
#define LOADLOCAL32(a, b)   asm volatile("{ ld.local.u32  %0, [%1]; }" : "=r"(a) : "l"(b))
#define LOADPARAM32(a, b)   asm volatile("{ ld.param.u32  %0, [%1]; }" : "=r"(a) : "l"(b))
#define LOADSHARED32(a, b)  asm volatile("{ ld.shared.u32 %0, [%1]; }" : "=r"(a) : "l"(b))

#define MOV32(a, b) 		asm volatile("{ mov.u32 %0, %1; }" : "=r"(a) : "r"(b))		
#define AND32(a, b) 		asm volatile("{ and.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		

#define OR32(a, b) 			asm volatile("{ or.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define XOR32(a, b) 		asm volatile("{ xor.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define NOT32(a, b) 		asm volatile("{ not.b32 %0, %1; }" : "=r"(a) : "r"(b))		
#define CNOT32(a, b) 		asm volatile("{ cnot.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
//#define LOP332(a, b) 		asm volatile("{ lop3.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define SHFLWRAP32(a, b) 	asm volatile("{ shf.l.wrap.b32 %0, %1, %2, %3; }" : "=r"(a) : "r"(b), "r"(a), "r"(b))		
#define SHFLCLAMP32(a, b) 	asm volatile("{ shf.l.clamp.b32 %0, %1, %2, %4; }" : "=r"(a) : "r"(b), "r"(a), "r"(b))		
#define SHL32(a, b) 		asm volatile("{ shl.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define SHR32(a, b) 		asm volatile("{ shr.b32 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		

#define ADD64(a, b) 		asm volatile("{ add.u64 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define ADDC64(a, b) 		asm volatile("{ addc.u64 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define ADDCCC64(a, b) 		asm volatile("{ addc.cc.u64 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define SUB64(a, b)    	 	asm volatile("{ sub.u64 %0, %1, %2; }" : "=r"(b) : "r"(b), "r"(a))		
#define MUL64(a, b)    	 	asm volatile("{ mul.lo.u64 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		
#define MAD64(a, b)    	 	asm volatile("{ mad.lo.u64 %0, %1, %2, %0; }" : "=r"(a) : "r"(b), "r"(a))		
#define MADCCC64(a, b) 		asm volatile("{ madc.lo.cc.u64 %0, %1, %2, %0; }" : "=r"(a) : "r"(b), "r"(a))		
#define MULADDCCC64(a, b)	asm volatile("{ mul.lo.u64 %0, %1, %2; addc.cc.u64 %0, %0, %0; }" : "=r"(a) : "r"(b), "r"(a))		
#define MOV64(a, b) 		asm volatile("{ mov.u64 %0, %1; }" : "=r"(a) : "r"(b))		
#define AND64(a, b) 		asm volatile("{ and.b64 %0, %1, %2; }" : "=r"(a) : "r"(b), "r"(a))		

/// TODO describe
#define INSTRUCTION_BENCH(OP, MULT) 																	\
__global__ 																								\
void kernel_instr_clock_##OP(uint32_t *ts, uint32_t *out, uint32_t p1, uint32_t p2, uint32_t its) {  	\
	uint32_t start_time = 0, stop_time = 0; 															\
	uint32_t tmp = p2; 																					\
	for (size_t i = 0; i < its; i++) { 																	\
		__syncthreads(); 																				\
		start_time = clock(); 																			\
		repeat##MULT(OP(tmp, p1); OP(p1, tmp);) 														\
		stop_time = clock(); 																			\
	} 																									\
	out[0] = tmp; 																						\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time; 											\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time; 											\
}

#define INSTRUCTION_MEM_BENCH(OP, MULT) 																\
__global__ 																								\
void kernel_instr_clock_##OP(uint32_t *ts, uint32_t *out, uint32_t p1, uint32_t p2, uint32_t its) {  	\
	uint32_t start_time = 0, stop_time = 0; 															\
	uint32_t tmp1 = p1, tmp2 = p2; 																		\
	for (size_t i = 0; i < its; i++) { 																	\
		__syncthreads(); 																				\
		start_time = clock(); 																			\
		repeat##MULT(OP(out[0], tmp2); OP(out[0], tmp1);) 													\
		stop_time = clock(); 																			\
	} 																									\
	out[0] += tmp1 + tmp2; 																				\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time; 											\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time; 											\
}
// TODO describe
#define INSTRUCTION_BENCH_WIDE(OP, MULT)																\
__global__ 																								\
void kernel_instr_clock_##OP(uint32_t *ts, uint32_t *out, uint32_t p1, uint32_t p2, uint32_t its) {  	\
	uint32_t start_time = 0, stop_time = 0; 															\
	uint32_t tmp = p2; uint64_t tmp2 = p1;																\
	for (size_t i = 0; i < its; i++) { 																	\
		__syncthreads(); 																				\
		start_time = clock(); 																			\
		repeat##MULT(OP(tmp2, uint32_t(tmp2), p1); OP(tmp2, tmp, uint32_t(tmp2));) 						\
		stop_time = clock(); 																			\
	} 																									\
	out[0] = tmp2 ^ (tmp2 >> 32u); 																		\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time; 											\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time; 											\
} 											

// TODO descrine
#define FUNCTION_BENCH_IMPL(OP, MULT, SIZE, T)												\
__global__ void kernel_function_clock_##OP(uint32_t *ts, T *out, T p1, T p2, uint32_t its){	\
	T t1[SIZE] = {0};																		\
	T t2[SIZE] = {0};																		\
	T t3[SIZE] = {0};																		\
	for (uint32_t i = 0; i < SIZE; i++) {													\
		t1[i] = p1; t2[i] = p2;	t3[i] = p1+p2;												\
	}																						\
	uint32_t start_time = 0, stop_time = 0; 												\
	for (uint32_t i = 0; i < its; i++) {													\
		__syncthreads();																	\
		start_time = clock();																\
		repeat##MULT(OP(t3, t1, t2); OP(t1, t2, t3); OP(t2, t3, t1);) 						\
		stop_time = clock();																\
	}																						\
	for (uint32_t i = 0; i < SIZE; i++) {													\
		out[i] = t1[i] + t2[i];																\
	}																						\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time;								\
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time;								\
}


#define FUNCTION_BENCH(OP, MULT, SIZE)   FUNCTION_BENCH_IMPL(OP,MULT,SIZE,uint32_t)
#define FUNCTION_BENCH64(OP, MULT, SIZE) FUNCTION_BENCH_IMPL(OP,MULT,SIZE,uint64_t)


// FUNC: kernel to call
// NUM:  number of threads in each Block
#define MEASURE_THROUGHPUT(FUNC, NUM, ITERS)											\
do {																					\
	Db.x = NUM; 																		\
																						\
	printf (" %s \tthroughput:\t", #FUNC);												\
	FUNC<<<Dg, Db>>>(d_ts, d_out, rand(), rand(), ITERS);								\
	hipDeviceSynchronize();															\
	hipError_t error;																	\
	if ((error = hipGetLastError()) != hipSuccess) {									\
		printf("  failed. %s\n\n", hipGetErrorString(error));							\
		break;																			\
	}																					\
	hipDeviceSynchronize();															\
	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);							\
	uint32_t min_t=uint32_t(-1), max_t=0;												\
	for (int i=0;i< Db.x*2;i++) {														\
		min_t = min(min_t, ts[i]);														\
		max_t = max(max_t, ts[i]);														\
	}																					\
																						\
	const double ops_clk = 	(Db.x*ITERS)/(double)(max_t-min_t); 						\
	printf("%9u clk (%.3f ops/clk, %.3f clk/ops), ", max_t-min_t, ops_clk, 1./ops_clk);	\
	printf("threads: %d, #ops: %d\n", NUM, ITERS);										\
} while (0)	

/// TODO preheating flag
/// Needs the following variables locally defined"
/// 	d_out, d_ts
/// FUNC: 		kernel to run
/// ITERS: 		nr of tries to averae over (32)
/// LTHREADS: 	number of threads to start (1)
/// HTHREADS: 	number of threads to end   (8192)
/// BITS:		number of bits in the argument of the kernel
#define PRINT_HISTOGRAM_IMPL(FUNC, ITERS, LTHREADS, HTHREADS, BITS)																\
do {																															\
	double tries = double(ITERS); 	 																							\
	printf ("\nPipeline latency/throughput with multiple warps (%d iterations of %d ops)\n", tries, ITERS);						\
	printf ("  %s:\n", #FUNC);																									\
	hipError_t code;  																											\
	uint32_t threads = LTHREADS, blocks = 1, tpb = 1;	  																		\
	for (; threads <= HTHREADS; threads += (threads < 4) ? 1 : (threads < 8) ? 2 : 												\
			(threads < 32) ? 8 : (threads < 128) ? 32 : (threads < 1024) ? 128: 1024) { 										\
		uint32_t histogram[1024] = {0};																							\
		uint32_t sum_time = 0, max_time, min_time, sum_max_time = 0; 															\
		bool failed = false;																									\
		blocks = (threads + 1023)/1024;  																						\
		tpb = threads >= 1024 ? 1024 : threads; 																				\
		for (uint32_t i=0; i < uint32_t(tries) && !failed; i++) {																\
			hipGetLastError();		/* Clear previous error code, if any */														\
			FUNC <<<blocks, tpb>>>(d_ts, d_out##BITS, rand(), rand(), iterations);												\
			code = hipGetLastError();  																						\
			if (code != hipSuccess) { 																							\
				failed = true;																									\
				break;																											\
			}																													\
																																\
			hipDeviceSynchronize();																							\
			hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);															\
																																\
			max_time = 0;																										\
			min_time = (unsigned)-1;																							\
			/* Compute histogram.	*/																							\
			for (int j=0; j< threads*2; j+= 64) {																				\
				sum_time += (ts[j+1] - ts[j]);																					\
				max_time = max(max_time, ts[j+1]);																				\
				min_time = min(min_time, ts[j]);																				\
				histogram[(ts[j+1] - ts[j])/ITERS]++;																			\
			}																													\
			sum_max_time += max_time-min_time;																					\
		}																														\
																																\
		if (failed) {																											\
			printf ("    %3d warp%c (%2d block%c, %4d tbp, %4d thread%c)  failed: %s", 											\
					(threads+31)/32, threads>=64 ? 's' : ' ', blocks, blocks > 1 ? 's' : ' ', tpb,    							\
					threads, threads > 1 ? 's' : ' ', hipGetErrorString(code));												\
		} else { 																												\
			/* Compute average latency over the lifetime of each warp (sum_time), 												\
			   and average throughput of the kernel (sum_max_time). */ 															\
			const double clk_warp = sum_time / tries / double(ITERS)/((threads+31)/32); 										\
			const double ops_clk  = ITERS*tries*threads/sum_max_time; 															\
			printf ("    %3d warp%c (%2d blk, %4d tpb, %4d thr) %9u clk (%.3f clk/warp, %.3f ops/clk, %.3f clk/ops) ", 			\
						(threads+31)/32, threads>=64 ? 's' : ' ', blocks, tpb, threads, sum_max_time,							\
						clk_warp, ops_clk, 1./ops_clk);																			\
		}																														\
																																\
		printf ("\n");																											\
	}																															\
																																\
    printf ("\n");																												\
} while(0)

//#define PRINT_HISTOGRAM(FUNC, ITERS)  	PRINT_HISTOGRAM_IMPL(FUNC, ITERS, 1, 8192, 32)
//#define PRINT_HISTOGRAM64(FUNC, ITERS)  PRINT_HISTOGRAM_IMPL(FUNC, ITERS, 1, 8192, 64)
#define PRINT_HISTOGRAM(FUNC, ITERS)  	PRINT_HISTOGRAM_IMPL(FUNC, ITERS, 1, 1024, 32)
#define PRINT_HISTOGRAM64(FUNC, ITERS)  PRINT_HISTOGRAM_IMPL(FUNC, ITERS, 1, 1024, 64)

#define MULTIPLIER 1024*2
//INSTRUCTION_MEM_BENCH(ATOMICADD32, 1024)
//INSTRUCTION_BENCH(ADD32, 1024)
//INSTRUCTION_BENCH(ADDC32, 1024)
//INSTRUCTION_BENCH(ADDCCC32, 1024)
//INSTRUCTION_BENCH(SUB32, 1024)
//INSTRUCTION_BENCH(MUL32, 1024)
//INSTRUCTION_BENCH_WIDE(MULWIDE, 1024)
//INSTRUCTION_BENCH(MAD32, 1024)
//INSTRUCTION_BENCH(MADCCC32, 1024)
//INSTRUCTION_BENCH(MULADDCCC32, 1024)
//INSTRUCTION_BENCH(MOV32, 1024)
//INSTRUCTION_BENCH(AND32, 1024)

//INSTRUCTION_BENCH(ADD64, 1024)
//INSTRUCTION_BENCH(ADDC64, 1024)
//INSTRUCTION_BENCH(ADDCCC64, 1024)
//INSTRUCTION_BENCH(SUB64, 1024)
//INSTRUCTION_BENCH(MUL64, 1024)
//INSTRUCTION_BENCH_WIDE(MULWIDE, 1024)
//INSTRUCTION_BENCH(MAD64, 1024)
//INSTRUCTION_BENCH(MADCCC64, 1024)
//INSTRUCTION_BENCH(MULADDCCC64, 1024)
//INSTRUCTION_BENCH(MOV64, 1024)
//INSTRUCTION_BENCH(AND64, 1024)

//FUNCTION_BENCH(fp_add96_32, 32, 3)
//FUNCTION_BENCH(fp_add128_32, 32, 4)
//FUNCTION_BENCH(fp_add160_32, 32, 5)
//FUNCTION_BENCH(fp_add192_32, 32, 6)
//FUNCTION_BENCH(fp_add224_32, 32, 7)
//FUNCTION_BENCH(fp_add256_32, 32, 8)

//FUNCTION_BENCH(fp_sub96_32, 32, 3)
//FUNCTION_BENCH(fp_sub128_32, 32, 4)
//FUNCTION_BENCH(fp_sub160_32, 32, 5)
//FUNCTION_BENCH(fp_sub192_32, 32, 6)
//FUNCTION_BENCH(fp_sub224_32, 32, 7)
//FUNCTION_BENCH(fp_sub256_32, 32, 8)

//FUNCTION_BENCH64(fp_add192_64, 32, 6)
//FUNCTION_BENCH64(fp_add256_64, 32, 6)
//FUNCTION_BENCH64(fp_sub192_64, 32, 6)
//FUNCTION_BENCH64(fp_sub256_64, 32, 6)


//FUNCTION_BENCH(karatsuba_mul2x2_32, 32, 4)
//FUNCTION_BENCH(karatsuba_mul2x2_32_v2, 32, 4)
//FUNCTION_BENCH(school_mul2x2_32, 32, 4)
//FUNCTION_BENCH(school_mul2x2_32_v6, 32, 4)
//FUNCTION_BENCH(school_mul2x2_32_v7, 32, 4)
//FUNCTION_BENCH(school_mul2x2_32_v2, 32, 4)
//FUNCTION_BENCH(school_mul2x2_32_v3, 32, 4)
//FUNCTION_BENCH(karatsuba_mul3x3_32, 32, 6)
//FUNCTION_BENCH(school_mul3x3_32, 32, 6)
//FUNCTION_BENCH(school_mul3x3_32_v2, 32, 6)
//FUNCTION_BENCH(karatsuba_mul4x4_32, 32, 8)
//FUNCTION_BENCH(school_sqr4x4_32, 32, 8)
//FUNCTION_BENCH(school_mul4x4_32, 32, 8)
//FUNCTION_BENCH(school_mul4x4_32_v7, 32, 8)
//FUNCTION_BENCH(school_mul4x4_32_v2, 32, 8)
//FUNCTION_BENCH(school_mul4x4_32_v3, 32, 8)
//FUNCTION_BENCH(school_mul5x5_32, 32, 10)
//FUNCTION_BENCH(karatsuba_mul6x6_32, 32, 12)
//FUNCTION_BENCH(school_mul6x6_32, 32, 12)
//FUNCTION_BENCH(karatsuba_mul7x7_32, 32, 14)
//FUNCTION_BENCH(school_mul7x7_32, 32, 14)
//FUNCTION_BENCH(karatsuba_mul8x8_32, 32, 16)
//FUNCTION_BENCH(school_mul8x8_32, 32, 16)


//FUNCTION_BENCH(school_mul3x3_low_32, 32, 3)
//FUNCTION_BENCH(school_mul3x3_low_32_v2, 32, 3)


//FUNCTION_BENCH(fp_karatsuba1x1_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x2_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x2_32_v2, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x3_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x4_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x5_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x6_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba1x7_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba2x2_32, 32, 4)
//FUNCTION_BENCH(fp_karatsuba3x3_32, 32, 3)
//FUNCTION_BENCH(fp_school3x3_32, 32, 3)
//FUNCTION_BENCH(fp_school3x3_32_v2, 32, 3)
//FUNCTION_BENCH(fp_school3x3_32_v3, 32, 3)
//FUNCTION_BENCH(fp_school3x3_32_v4, 32, 3)
//FUNCTION_BENCH(fp_school3x3_32_v5, 32, 3)
//FUNCTION_BENCH(fp_karatsuba4x4_32, 32, 4)
//FUNCTION_BENCH(fp_school4x4_32, 32, 4)
//FUNCTION_BENCH(fp_school4x4_32_v2, 32, 4)
//FUNCTION_BENCH(fp_school4x4_32_v3, 32, 4)
//FUNCTION_BENCH(fp_school4x4_32_v7, 32, 4)
//FUNCTION_BENCH(fp_sqr4x4_32, 32, 4)
//FUNCTION_BENCH(fp_school5x5_32, 32, 5)
FUNCTION_BENCH(fp_karatsuba6x6_32, 32, 6)
FUNCTION_BENCH(fp_school6x6_32, 32, 6)
//FUNCTION_BENCH(fp_karatsuba7x7_32, 32, 7)
//FUNCTION_BENCH(fp_school7x7_32, 32, 7)
//FUNCTION_BENCH(fp_karatsuba8x8_32, 32, 8)
//FUNCTION_BENCH(fp_school8x8_32, 32, 8)

//FUNCTION_BENCH64(fp_karatsuba2x2_64, 64, 2)
//FUNCTION_BENCH64(fp_karatsuba3x3_64, 64, 3)
FUNCTION_BENCH64(fp_karatsuba4x4_64, 64, 4)


//FUNCTION_BENCH64(school_mul1x1_64_v2, 64, 2)
//FUNCTION_BENCH64(school_mul1x1_64_v3, 64, 2)
//FUNCTION_BENCH64(school_mul2x2_64, 64, 4)
//FUNCTION_BENCH64(school_mul2x2_64_v2, 64, 4)


	__constant__ uint32_t constloadbenchdata[100*32] = {1};
__global__ uint32_t globalloadbenchdata[100*32] = {1};
__global__ void kernel_instr_clock_LOAD32(uint32_t *ts, uint32_t *out, uint32_t *extdata, uint32_t its){
	uint32_t data = 0, tmp;
	uint32_t start_time = 0, stop_time = 0;
	const size_t index = threadIdx.x + blockIdx.x*blockDim.x;
	const size_t blocksize = blockIdx.x*blockDim.x;

	for (uint32_t i = 0; i < its; i++) {
		__syncthreads();
		start_time = clock();
		repeat128(LOADCONST32(tmp, extdata+i*blocksize+index); ADD32(data, tmp);)
		//LOADCONST32(tmp, extdata+i+index);
		stop_time = clock();
		data += tmp;
	}

	out[0] = data;
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2] = start_time;
	ts[(blockIdx.x*blockDim.x + threadIdx.x)*2+1] = stop_time;
}

void print_information(uint32_t *ts, const dim3 Dg, const uint32_t mult, const uint32_t empty_cycles){
	int m = 0;
	const uint32_t off = 1;
	const double MULT = mult;
	for (uint32_t i = 0; i < off; i++) {	
		for (uint32_t j = 0; j < Dg.x/off; j++, m++) {
			uint32_t start = ts[(off*j+i)*2];
			uint32_t end   = ts[(off*j+i)*2+1];
			printf("  Block %02d: start: %08u, stop: %08u, diff: %u, clock: %lf\n", off*j+i, start, end, end-start, double(end-start)/MULT);
		}
	}	
}

int main() {
	srand(time(NULL));
	uint32_t ts[1024], out32[128];
	uint32_t *d_ts,*d_ts2,*d_out32;

	uint64_t out64[128];
	uint64_t *d_out64;
	
	dim3 Db = dim3(1);
    dim3 Dg = dim3(1,1,1);
	
	Dg.x = 1; 
	Db.x = 32;

	// Allocate device array.
	hipError_t errcode;
	if (hipSuccess != (errcode = hipMalloc((void**)&d_ts, sizeof(ts)))) {
		printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
		printf ("   %s\n", hipGetErrorString(errcode));
		return;
	}
	if (hipSuccess != (errcode = hipMalloc((void**)&d_ts2, sizeof(ts)))) {
		printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
		printf ("   %s\n", hipGetErrorString(errcode));
		return;
	}
	if (hipSuccess != (errcode = hipMalloc((void**)&d_out32, 4 * 128))) {
		printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
		printf ("   %s\n", hipGetErrorString(errcode));
		return;
	}
	if (hipSuccess != (errcode = hipMalloc((void**)&d_out64, 8 * 128))) {
		printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
		printf ("   %s\n", hipGetErrorString(errcode));
		return;
	}

	// clear any error
	hipGetLastError();
	printf ("\nkclock (preheating): ");
	for (uint64_t i = 0; i < 1000; i++) {
		kclock <<<Dg, Db>>> (d_ts);
		hipDeviceSynchronize();
	}
	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);

	printf("\n");
	//for (int i=0; i< Dg.x*Db.x; i++)
	//	printf ("   (%8u, %8u): %u\n", ts[i*2], ts[i*2+1], ts[i*2+1]-ts[i*2]);
    //	printf ("\n");	

	// meassures the number of cycles for doing nothing
	const uint32_t empty_cycles = ts[1]-ts[0];
	const uint32_t iterations = 16;
	
//	printf ("\nkernel_instr_clock_LOADCONST32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_instr_clock_LOAD32<<<Dg, Db>>>(d_ts, d_out, constloadbenchdata, 2);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 128, empty_cycles);
//
//	printf ("\nkernel_instr_clock_LOADGLOBAL32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_instr_clock_LOAD32<<<Dg, Db>>>(d_ts, d_out, globalloadbenchdata, 2);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 128, empty_cycles);
//
//	printf ("\nkernel_instr_clock_ADD32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_instr_clock_ADD32<<<Dg, Db>>>(d_ts, d_out, 268324234u, 876218736u, iterations);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 2*128, empty_cycles);

//	printf ("\nkernel_instr_clock_ATOMICADD32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_instr_clock_ATOMICADD32<<<Dg, Db>>>(d_ts, d_out, 268324234u, 876218736u, iterations);
//	hipGetLastError();
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 2*128, empty_cycles);


//	MEASURE_THROUGHPUT(kernel_instr_clock_ADD32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_ATOMICADD32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_ADDC32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_ADD32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_SUB32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_MUL32, 32, 1024*2);
//  MEASURE_THROUGHPUT(kernel_instr_clock_MULWIDE, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_MAD32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_MADCCC32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_MULADDCCC32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_MOV32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_instr_clock_AND32, 32, 1024*2);

// 	PRINT_HISTOGRAM(kernel_instr_clock_ADD32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_ATOMICADD32, 1024*2);
//  PRINT_HISTOGRAM(kernel_instr_clock_ADDC32, 1024*2);
//  PRINT_HISTOGRAM(kernel_instr_clock_ADDCCC32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_SUB32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_MUL32, 1024*2);
//  PRINT_HISTOGRAM(kernel_instr_clock_MULWIDE, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_MAD32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_MADCCC32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_MULADDCCC32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_MOV32, 1024*2);
//	PRINT_HISTOGRAM(kernel_instr_clock_AND32, 1024*2);


//	PRINT_HISTOGRAM(kernel_function_clock_fp_add96_32,  3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_add128_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_add160_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_add192_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_add224_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_add256_32, 3*32);
	
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub96_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub128_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub160_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub192_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub224_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sub256_32, 3*32);


//	PRINT_HISTOGRAM64(kernel_function_clock_fp_add192_64, 3*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_fp_add256_64, 3*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_fp_sub192_64, 3*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_fp_sub256_64, 3*32);

//	MEASURE_THROUGHPUT(kernel_function_clock_fp_karatsuba1x2_32, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_function_clock_fp_karatsuba1x2_32_v2, 32, 1024*2);
//	MEASURE_THROUGHPUT(kernel_function_clock_fp_karatsuba1x2_32, 128, 1024*2);
//	MEASURE_THROUGHPUT(kernel_function_clock_fp_karatsuba1x2_32_v2, 128, 1024*2);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba1x2_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba1x2_32_v2, 3*32);

//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul2x2_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul2x2_32_v2, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul2x2_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul2x2_32_v6, 32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul2x2_32_v7, 32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul2x2_32_v2, 32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul2x2_32_v3, 32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul3x3_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul3x3_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul3x3_32_v2, 32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul4x4_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul4x4_32,    32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul4x4_32_v7, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul4x4_32_v2, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul4x4_32_v3, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_sqr4x4_32,    32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul5x5_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul6x6_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul6x6_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul7x7_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul7x7_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_karatsuba_mul8x8_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul8x8_32,    32*32);
	

//	PRINT_HISTOGRAM(kernel_function_clock_school_mul3x3_low_32,    32*32);	
//	PRINT_HISTOGRAM(kernel_function_clock_school_mul3x3_low_32_v2, 32*32);	


//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba3x3_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school3x3_32,    32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school3x3_32_v2, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school3x3_32_v3, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school3x3_32_v4, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school3x3_32_v5, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba4x4_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school4x4_32,    32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school4x4_32_v2, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school4x4_32_v3, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school4x4_32_v7, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_sqr4x4_32,       32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school5x5_32,    32*32);
	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba6x6_32, 32*32);
	PRINT_HISTOGRAM(kernel_function_clock_fp_school6x6_32,    32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba7x7_32, 32*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school7x7_32,    32*32);
	
//	PRINT_HISTOGRAM(kernel_function_clock_fp_karatsuba_mul8x8_32, 3*32);
//	PRINT_HISTOGRAM(kernel_function_clock_fp_school_mul8x8_32, 3*32);


//	PRINT_HISTOGRAM64(kernel_function_clock_fp_karatsuba2x2_64, 32*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_fp_karatsuba3x3_64, 32*32);
	PRINT_HISTOGRAM64(kernel_function_clock_fp_karatsuba4x4_64, 32*32);

//	PRINT_HISTOGRAM64(kernel_function_clock_school_mul1x1_64_v2, 32*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_school_mul1x1_64_v3, 32*32);
	
//	PRINT_HISTOGRAM64(kernel_function_clock_school_mul2x2_64, 32*32);
//	PRINT_HISTOGRAM64(kernel_function_clock_school_mul2x2_64_v2, 32*32);

//  printf ("\nkernel_function_clock_fp_karatsuba_mul128_32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//  kernel_function_clock_fp_karatsuba_mul128_32<<<Dg, Db>>>(d_ts, d_out, 268324234u, 876218736u, iterations);
//  hipDeviceSynchronize();
//  hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//  print_information(ts, Dg, 3*128, empty_cycles);

//	printf ("\nkernel_clock_fpadd32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_clock_fpadd32<<<Dg, Db>>>(d_ts, d_out, 268324234u, 876218736u, iterations);
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 3*128, empty_cycles);
//
//	printf ("\nkernel_clock_fpadd64: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_clock_fpadd<<<Dg, Db>>>(d_ts, d_out, 198743268324234ull, 876218736ull, iterations);
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 3*128, empty_cycles);
//
//	printf ("\nkernel_clock_fpmul32: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_clock_fpmul32<<<Dg, Db>>>(d_ts, d_out, 268324234u, 876218736u, iterations);
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 3*128, empty_cycles);
//
//	printf ("\nkernel_clock_fpmul64: [%d blocks, %d thread(s)/block]\n", Dg.x, Db.x);
//	kernel_clock_fpmul<<<Dg, Db>>>(d_ts, d_out, 198743268324234ull, 876218736ull, iterations);
//	hipDeviceSynchronize();
//	hipMemcpy(ts, d_ts, sizeof(ts), hipMemcpyDeviceToHost);
//	print_information(ts, Dg, 3*128, empty_cycles);

	// make some how sure that the compile does not throw away the instruction, 
	// want to benchmark.
	hipDeviceSynchronize();
	hipMemcpy(out32, d_out32, sizeof(out32), hipMemcpyDeviceToHost);
	hipMemcpy(out64, d_out64, sizeof(out64), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipFree (d_ts);
	hipFree (d_ts2);
	return 0;
}
