#include "hip/hip_runtime.h"
#include <stdlib.h>

#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include "karatsuba64.cu"
#include "arith.cuh"

//__device__ const uint64_t __p[2]   = { 0xAC0E7A06FFFFFFFD, 0x0000000000000012 };
//__device__ const uint32_t __p32[4] = { 0xFFFFFFFD, 0xAC0E7A06, 0x00000012, 0x0 };
//
//__device__ const uint64_t __mu[2]   = { 0x1, 0x0000000000000012 };
//__device__ const uint32_t __mu32[4] = { 0x1, 0x1, 0x1, 0x0 };

//__constant__ limb_t __p[NWORDS_FIELD] = { 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x7BC6BFFF, 0x8AF43417, 0xA1087212, 0x01248A1B };
//__constant__ limb_t __mu[NWORDS_FIELD] = { 0x00000001, 0x00000000, 0x00000000, 0x7BC6C000, 0x8AF43417, 0xA1087212, 0x91248A1B };

// this are the 6 limb p and mu value
//__constant__ uint32_t __p[8]  = { 0xFFFFFFFF, 0xFFFFFFFF, 0x17FFFFFF, 0xD9A6C7B1, 0x20D5040B, 0x002895AC, 0, 0 };
//__constant__ uint32_t __mu[8] = { 0x00000001, 0x00000000, 0x18000000, 0xD9A6C7B1, 0x20D5040B, 0x326895AC, 0, 0 };

// these are the 8 limb p and mu value 
__constant__ uint32_t __p[8] = { 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x7BC6BFFF, 0x8AF43417, 0xA1087212, 0x01248A1B, 0 };
__constant__ uint32_t __mu[8] = { 0x00000001, 0x00000000, 0x00000000, 0x7BC6C000, 0x8AF43417, 0xA1087212, 0x91248A1B, 0 };

// just testing
//uint32_t __cpu_p[8]  = { 0xFFFFFFFF, 0xFFFFFFFF, 0x17FFFFFF, 0xD9A6C7B1, 0x20D5040B, 0x002895AC, 0, 0 };
//uint32_t __cpu_mu[8] = { 0x00000001, 0x00000000, 0x18000000, 0xD9A6C7B1, 0x20D5040B, 0x326895AC, 0, 0 };
uint32_t __cpu_p[8] = { 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x7BC6BFFF, 0x8AF43417, 0xA1087212, 0x01248A1B };
uint32_t __cpu_mu[8] = { 0x00000001, 0x00000000, 0x00000000, 0x7BC6C000, 0x8AF43417, 0xA1087212, 0x91248A1B };





// first part of montgomery reduction 
// essential its a multiplication a*mu mod 2**64
// This version does not use the 'mad' instructions
#define __school32_inplace_2x2_low_v2(a0,a1,mu0,mu1)\
	asm volatile ("{\n\t"                       \
			".reg .u32 t10;\n\t"                \
			/* t0-t2: m = a * mu mod 2^64 */    \
			/* Multiplication involving a0 */   \
			"mul.lo.u32     %0,  %4, %2;\n\t"	\
			"mul.hi.u32     %1,  %4, %2;\n\t"   \
			"mul.lo.u32     t10, %4, %3;\n\t"   \
			/* now fixing the carry chains */   \
  			"add.cc.u32     %1,  %1,  t10;\n\t" \
			/*Multiplication involving a3 second (low) diagonal*/   		\
			"mul.lo.u32     t10, %5,  %3;\n\t"  \
			"add.u32         %1, %1, t20;\n\t"  \
			"}\n"		                          		\
			: "=r"(a0),"=r"(a1),"=r"(a2)   /*0 , 1*/ 	\
			: "r"(mu0),"r"(mu1),"r"(mu2)   /*2 , 3*/ 	\
			  "r"(a0),"r"(a1),"r"(a2)  	   /*4 , 5*/	\
		);


// second part of the montgomery reduction written by floyd
#define __reduce32_sec10(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,p0,p1,p2,p3,p4,i0,i1,i2,i3,i4,i5,i6,i7,i8,i9)	 	\
	asm volatile ("{\n\t"                   						\
			/*%0-%5: r= (a + u) div 2^(5**32)*/      				\
			".reg.u32 		t0, t5,t6,t7,t8,t9;\n\t"				\
			"mov.b32        t0, 0;\n\t"             				\
			/*a + u*/												\
			"add.cc.u32     %0, %20, %10;\n\t"        				\
			"addc.cc.u32    %1, %21, %11;\n\t"        				\
			"addc.cc.u32    %2, %22, %12;\n\t"        				\
			"addc.cc.u32    %3, %23, %13;\n\t"        				\
			"addc.cc.u32    %4, %24, %14;\n\t"       				\
			"addc.cc.u32    t5, %25, %15;\n\t"       				\
			"addc.cc.u32    t6, %26, %16;\n\t"       				\
			"addc.cc.u32    t7, %27, %17;\n\t"       				\
			"addc.cc.u32    t8, %28, %18;\n\t"       				\
			"addc.cc.u32    t9, %29, %19;\n\t"       				\
			"addc.u32       t0, t0, 0;\n\t"         				\
			/*-p*/													\
			"sub.cc.u32     t5, t5, %5;\n\t"       					\
			"subc.cc.u32    t6, t6, %6;\n\t"       					\
			"subc.cc.u32    t7, t7, %7;\n\t"       					\
			"subc.cc.u32    t8, t8, %8;\n\t"       					\
			"subc.cc.u32    t9, t9, %9;\n\t"       					\
			"subc.u32       t0, t0, 0;\n\t"         				\
			/*&p*/													\
			"and.b32        %0, t0, %5;\n\t"       					\
			"and.b32        %1, t0, %6;\n\t"       					\
			"and.b32        %2, t0, %7;\n\t"       					\
			"and.b32        %3, t0, %8;\n\t"       					\
			"and.b32        %4, t0, %9;\n\t"       					\
			"add.cc.u32     %0, %0, t5;\n\t"        				\
			"addc.cc.u32    %1, %1, t6;\n\t"        				\
			"addc.cc.u32    %2, %2, t7;\n\t"        				\
			"addc.cc.u32    %3, %3, t8;\n\t"        				\
			"addc.u32       %4, %4, t9;\n\t"       					\
			"}\n"                                     				\
	: "=r"(a0),"=r"(a1),"=r"(a2),"=r"(a3),"=r"(a4)	/*0 , 5*/ 		\
	: "r"(p0),"r"(p1),"r"(p2),"r"(p3),"r"(p4),	 	/*5 , 9*/  		\
	  "r"(i0),"r"(i1),"r"(i2),"r"(i3),"r"(i4), 		/*10,14*/  		\
	  "r"(i5),"r"(i6),"r"(i7),"r"(i8),"r"(i9),      /*15,19*/  		\
	  "r"(a0),"r"(a1),"r"(a2),"r"(a3),"r"(a4), 		/*20,24*/  		\
	  "r"(a5),"r"(a6),"r"(a7),"r"(a8),"r"(a9)       /*25,29*/  		\
	);


// montgomery reuction for 10 32bit limbs. 
#define __reduce32_inplace10(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,mu0,mu1,mu2,mu3,mu4,p0,p1,p2,p3,p4)		\
	uint32_t t0,t1,t2,t2,t3,t4,t5,t6,t7,t8,t9;														\
	__school32_inplace_5x5_low_v2(a0,a1,a2,a3,a4,mu0,mu1,mu2,mu3,mu4)								\
	__school32_5x5(t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,a0,a1,a2,a3,a4,p0,p1,p2,p3,p4)   					\
	__reduce32_sec10(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,p0,p1,p2,p3,p4,t0,t1,t2,t3,t4,t5,t6,t7,t8,t9)


// TODO describe
#define __reduce32_sec16(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,a10,a11,a12,a13,a14,a15, p0,p1,p2,p3,p4,p5,p6,p7, i0,i1,i2,i3,i4,i5,i6,i7,i8,i9,i10,i11,i12,i13,i14,i15)	 	 \
        asm volatile ("{\n\t"						\
			/* r = (a + u) div 2^(8*32)*/      		\
       		 ".reg .u32		 t0, t8,t9,t10,t11,t12,t13,t14,t15;\n\t"\
			 "mov.u32        t0, 0;\n\t"            \
       		 "add.cc.u32     %0, %32, %16;\n\t"     \
       		 "addc.cc.u32    %1, %33, %17;\n\t"     \
       		 "addc.cc.u32    %2, %34, %18;\n\t"     \
       		 "addc.cc.u32    %3, %35, %19;\n\t"     \
       		 "addc.cc.u32    %4, %36, %20;\n\t"     \
       		 "addc.cc.u32    %5, %37, %21;\n\t"     \
       		 "addc.cc.u32    %6, %38, %22;\n\t"     \
       		 "addc.cc.u32    %7, %39, %23;\n\t"     \
       		 "addc.cc.u32    t8, %40, %24;\n\t"     \
       		 "addc.cc.u32    t9, %41, %25;\n\t"     \
       		 "addc.cc.u32    t10, %42, %26;\n\t"    \
       		 "addc.cc.u32    t11, %43, %27;\n\t"    \
       		 "addc.cc.u32    t12, %44, %28;\n\t"    \
       		 "addc.cc.u32    t13, %45, %29;\n\t"    \
       		 "addc.cc.u32    t14, %46, %30;\n\t"    \
       		 "addc.cc.u32    t15, %47, %31;\n\t"    \
       		 "addc.u32       t0, t0, 0;\n\t"        \
       		 "sub.cc.u32     t8, t8, %8;\n\t"       \
       		 "subc.cc.u32    t9, t9, %9;\n\t"       \
       		 "subc.cc.u32    t10, t10, %10;\n\t"    \
       		 "subc.cc.u32    t11, t11, %11;\n\t"    \
       		 "subc.cc.u32    t12, t12, %12;\n\t"    \
       		 "subc.cc.u32    t13, t13, %13;\n\t"    \
       		 "subc.cc.u32    t14, t14, %14;\n\t"    \
       		 "subc.cc.u32    t15, t15, %15;\n\t"    \
			 "subc.u32       t0, t0, 0;\n\t"        \
       		 "and.b32        %0, t0, %8;\n\t"       \
       		 "and.b32        %1, t0, %9;\n\t"       \
       		 "and.b32        %2, t0, %10;\n\t"      \
       		 "and.b32        %3, t0, %11;\n\t"      \
       		 "and.b32        %4, t0, %12;\n\t"      \
       		 "and.b32        %5, t0, %13;\n\t"      \
       		 "and.b32        %6, t0, %14;\n\t"      \
       		 "and.b32        %7, t0, %15;\n\t"      \
       		 "add.cc.u32     %0, %0, t8;\n\t"       \
       		 "addc.cc.u32    %1, %1, t9;\n\t"       \
       		 "addc.cc.u32    %2, %2, t10;\n\t"      \
       		 "addc.cc.u32    %3, %3, t11;\n\t"      \
       		 "addc.cc.u32    %4, %4, t12;\n\t"      \
       		 "addc.cc.u32    %5, %5, t13;\n\t"      \
       		 "addc.cc.u32    %6, %6, t14;\n\t"      \
       		 "addc.u32       %7, %7, t15;\n\t"      \
        "}\n\t"                                     \
        : "=r"(a0),"=r"(a1),"=r"(a2),"=r"(a3),"=r"(a4),"=r"(a5),"=r"(a5),"=r"(a7) 					/*0 - 7*/\
        : "r"(p0), "r"(p1), "r"(p2), "r"(p3), "r"(p4), "r"(p5), "r"(p6), "r"(p7), 					/*8 -15*/\
          "r"(i0), "r"(i1), "r"(i2), "r"(i3), "r"(i4), "r"(i5), "r"(i6), "r"(i7),               	/*16-23*/\
          "r"(i8), "r"(i9), "r"(i10), "r"(i11), "r"(i12), "r"(i13), "r"(i14), "r"(i15),				/*24-31*/\
          "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(a4), "r"(a5), "r"(a6), "r"(a7),               	/*32-39*/\
          "r"(a8), "r"(a9), "r"(a10), "r"(a11), "r"(a12), "r"(a13), "r"(a14), "r"(a15) 				/*40-47*/\
		  )


// TODO describe
#define __reduce32_inplace16(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,a10,a11,a12,a13,a14,a15,mu0,mu1,mu2,mu3,mu4,mu5,mu6,mu7,p0,p1,p2,p3,p4,p5,p6,p7)	\
	uint32_t t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15;																	\
	__school32_inplace_8x8_low_v2(a0,a1,a2,a3,a4,a5,a6,a7,mu0,mu1,mu2,mu3,mu4,mu5,mu6,mu7) \
	__school32_8x8(t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15,a0,a1,a2,a3,a4,a5,a6,a7,p0,p1,p2,p3,p4,p5,p6,p7) 			\
	__reduce32_sec16(a0,a1,a2,a3,a4,a5,a6,a7,a8,a9,a10,a11,a12,a13,a14,a15, p0,p1,p2,p3,p4,p5,t6,p7, t0,t1,t2,t3,t4,t5,t6,t8,t8,t9,t10,t11,t12,t13,t14,t15)



// TODO describe
#define __reduce32_16(a0,a1,a2,a3,a4,a5,a6,a7, c0,c1,c2,c3,c4,c5,c6,c7,c8,c9,c10,c11,c12,c13,c14,c15, mu0,mu1,mu2,mu3,mu4,mu5,mu6,mu7, p0,p1,p2,p3,p4,p5,p6,p7)	\
//

// TODO describe
#define __fp_karatsuba_mul128_32(c0,c1,c2,c3,a0,a1,a2,a3,b0,b1,b2,b3,m0,m1,m2,m3,p0,p1,p2,p3)  	\
	uint32_t t0,t1,t2,t3,t4,t5,t6,t7; 															\
	__karatsuba_4x4(t0,t1,t2,t3,t4,t5,t6,t7, a0,a1,a2,a3, b0,b1,b2,b3)							\
	__reduce8(c0,c1,c2,c3, t0,t1,t2,t3,t4,t5,t6,t7, m0,m1,m2,m3, p0,p1,p2,p3)

// TODO describe
#define __fp_karatsuba_mul192_32(c0,c1,c2,c3,c4,c5,a0,a1,a2,a3,a4,a5,b0,b1,b2,b3,b4,b5,m0,m1,m2,m3,m4,m5,p0,p1,p2,p3,p4,p5)  	\
	uint32_t t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11; 																			\
	__karatsuba_6x6(t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11, a0,a1,a2,a3,a4,a5, b0,b1,b2,b3,b4,b5)								\
	__reduce12(c0,c1,c2,c3,c4,c5, t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11, m0,m1,m2,m3,m4,m5, p0,p1,p2,p3,p4,p5)

// TODO describe:
#define __fp_karatsuba_mul224_32(c0,c1,c2,c3,c4,c5,c6,a0,a1,a2,a3,a4,a5,a6,b0,b1,b2,b3,b4,b5,b6,m0,m1,m2,m3,m4,m5,m6,p0,p1,p2,p3,p4,p5,p6)  	\
	uint32_t t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13; 																						\
	__karatsuba_7x7(t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13, a0,a1,a2,a3,a4,a5,a6, b0,b1,b2,b3,b4,b5,b6)								\
	__reduce16(c0,c1,c2,c3,c4,c5,c6,c7, t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14, m0,m1,m2,m3,m4,m5,m6, p0,p1,p2,p3,p4,p5,p6) 	

// TODO describe:
#define __fp_karatsuba_mul256_32(c0,c1,c2,c3,c4,c5,c6,c7,a0,a1,a2,a3,a4,a5,a6,a7,b0,b1,b2,b3,b4,b5,b6,b7,m0,m1,m2,m3,m4,m5,m6,m7,p0,p1,p2,p3,p4,p5,p6,p7)  	\
	uint32_t t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15; 																						\
	__karatsuba_8x8(t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15, a0,a1,a2,a3,a4,a5,a6,a7, b0,b1,b2,b3,b4,b5,b6,b7)								\
	__reduce16(c0,c1,c2,c3,c4,c5,c6,c7, t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15, m0,m1,m2,m3,m4,m5,m6,m7, p0,p1,p2,p3,p4,p5,p6,p7) 	

//__global__ void simple_test1(uint32_t *out, uint32_t p1){
//	uint32_t t1[2] = {p1, 0};
//	uint32_t t2[2] = {p1, p1};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_2x2(out[0], out[1], out[2], out[3], t1[0], t1[1], t2[0], t2[1]);
//}
//
//__global__ void simple_test2(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[2] = {p1, p1};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x2(out[0], out[1], out[2], t1[0], t2[0], t2[1]);
//}
//
//__global__ void simple_test2_v2(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[2] = {p1, p1};
//	for (size_t i = 0; i < 16; i++) { out[i] = 0; }
//	__karatsuba32_1x2_v2(out[0], out[1], out[2], t1[0], t2[0], t2[1]);
//}
//
//__global__ void simple_test3(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[3] = {1, 0, 0};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x3(out[0], out[1], out[2], out[3], t1[0], t2[0], t2[1], t2[2]);
//}
//
//__global__ void simple_test4(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[4] = {1, 0, 0, 0};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x4(out[0], out[1], out[2], out[3], out[4], t1[0], t2[0], t2[1], t2[2], t2[3]);
//}
//
//__global__ void simple_test5(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[5] = {1, 0, 0, 0, 0};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x5(out[0], out[1], out[2], out[3], out[4], out[5], t1[0], t2[0], t2[1], t2[2], t2[3], t2[4]);
//}
//
//__global__ void simple_test6(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[6] = {1, 0, 0, 0, 0, 0};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x6(out[0], out[1], out[2], out[3], out[4], out[5], out[6], t1[0], t2[0], t2[1], t2[2], t2[3], t2[4], t2[5]);
//}
//
//__global__ void simple_test7(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[7] = {1, 0, 0, 0, 0, 0, 0};
//	for (size_t i = 0; i < 8; i++) { out[i] = 0; }
//	__karatsuba32_1x7(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], t1[0], t2[0], t2[1], t2[2], t2[3], t2[4], t2[5], t2[6]);
//}
//
//__global__ void simple_test8(uint32_t *out, uint32_t p1){
//	uint32_t t1[1] = {p1};
//	uint32_t t2[8] = {1, 0, 0, 0, 0, 0, 0, 0};
//	for (size_t i = 0; i < 9; i++) { out[i] = 0; }
//	__karatsuba32_1x8(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], out[8], t1[0], t2[0], t2[1], t2[2], t2[3], t2[4], t2[5], t2[6], t2[7]);
//}
//
//__global__ void simple_test6444(uint64_t *out, uint64_t p1){
//	uint64_t t1[4] = {p1, 0, 0, 0};
//	uint64_t t2[4] = {1, 0, 0, 0};
//	for(uint64_t i = 0; i < 8; i++){out[i] = 0;}
//	__karatsuba64_4x4(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], t1[0], t1[1], t1[2], t1[3], t2[0], t2[1], t2[2], t2[3]);
//}
//__global__ void simple_test44(uint32_t *out, uint32_t p1){
//	uint32_t t1[4] = {p1, 0, 0, 0};
//	uint32_t t2[4] = {1, 0, 0, 0};
//	for(uint32_t i = 0; i < 8; i++){out[i] = 0;}
//	__karatsuba32_4x4(out[0], out[1], out[2], out[3], out[4], out[5], out[6], out[7], t1[0], t1[1], t1[2], t1[3], t2[0], t2[1], t2[2], t2[3]);
//}
//
//__global__ void simple_test6433(uint64_t *out, uint64_t p1){
//	uint64_t t1[3] = {p1, 0, 0};
//	uint64_t t2[3] = {1, 0, 0};
//	for(uint64_t i = 0; i < 8; i++){out[i] = 0;}
//	__karatsuba64_3x3(out[0], out[1], out[2], out[3], out[4], out[5], t1[0], t1[1], t1[2], t2[0], t2[1], t2[2]);
//}
//__global__ void simple_test33(uint32_t *out, uint32_t p1){
//	uint32_t t1[3] = {p1, 0, 0};
//	uint32_t t2[3] = {1, 0, 0};
//	for(uint32_t i = 0; i < 8; i++){out[i] = 0;}
//	__karatsuba32_3x3(out[0], out[1], out[2], out[3], out[4], out[5], t1[0], t1[1], t1[2], t2[0], t2[1], t2[2]);
//}
//
//__global__ void simple_test6x6(uint32_t *out, uint32_t p1){
//	uint32_t t1[6] = {p1,0,0,0,0,0};
//	uint32_t t2[6] = {1,0,0,0,0,0};
//	for(uint32_t i = 0; i < 12; i++){out[i] = 0;}
//	__karatsuba32_6x6(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5]);
//}
//
//__global__ void simple_test8x8(uint32_t *out, uint32_t p1){
//	uint32_t t1[8] = {p1,0,0,0,0,0,0,0};
//	uint32_t t2[8] = {1,0,0,0,0,0,0,0};
//	for(uint32_t i = 0; i < 16; i++){out[i] = 0;}
//	__karatsuba32_8x8(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13],out[14],out[15], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5],t1[6],t1[7], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5],t2[6],t2[7]);
//}
//
//__global__ void simple_test_school8x8(uint32_t *out, uint32_t p1){
//	uint32_t t1[8] = {p1,0,0,0,0,0,0,0};
//	uint32_t t2[8] = {1,0,0,0,0,0,0,0};
//	for(uint32_t i = 0; i < 16; i++){out[i] = 0;}
//	__school32_8x8(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13],out[14],out[15], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5],t1[6],t1[7], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5],t2[6],t2[7]);
//}
//__global__ void simple_test_school7x7(uint32_t *out, uint32_t p1){
//	uint32_t t1[7] = {p1,0,0,0,0,0,0};
//	uint32_t t2[7] = {1,0,0,0,0,0,0};
//	for(uint32_t i = 0; i < 16; i++){out[i] = 0;}
//	__school32_7x7(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5],t1[6], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5],t2[6]);
//}
//__global__ void simple_test_school6x6(uint32_t *out, uint32_t p1){
//	uint32_t t1[6] = {p1,0,0,0,0,0};
//	uint32_t t2[6] = {1,0,0,0,0,0};
//	for(uint32_t i = 0; i < 16; i++){out[i] = 0;}
//	__school32_6x6(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5]);
//}
//
//__global__ void simple_test_fp_school6x6(uint32_t *out, uint32_t p1){
//	uint32_t t1[6] = {p1,0,0,0,0,0};
//	uint32_t t2[6] = {1,0,0,p1,0,0};
//	for(uint32_t i = 0; i < 6; i++){out[i] = 0;}
//	__school32_6x6(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], t1[0],t1[1],t1[2],t1[3],t1[4],t1[5], t2[0],t2[1],t2[2],t2[3],t2[4],t2[5]);
//	__reduce32_6x6(out[0],out[1],out[2],out[3],out[4],out[5], out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], __mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5], __p[0],__p[1],__p[2],__p[3],__p[4],__p[5]); 
//}
//
//
//__global__ void simple_test_school_inplace_3x3_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_3x3_low(out[0],out[1],out[2], __mu[0],__mu[1],__mu[2]);
//}
//__global__ void simple_test_school_inplace_4x4_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_4x4_low(out[0],out[1],out[2],out[3], __mu[0],__mu[1],__mu[2],__mu[3]);
//}
//__global__ void simple_test_school_inplace_5x5_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_5x5_low(out[0],out[1],out[2],out[3],out[4], __mu[0],__mu[1],__mu[2],__mu[3],__mu[4]);
//}
//__global__ void simple_test_school_inplace_6x6_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_6x6_low(out[0],out[1],out[2],out[3],out[4],out[5], __mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5]);
//}
//__global__ void simple_test_school_inplace_7x7_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_7x7_low(out[0],out[1],out[2],out[3],out[4],out[5],out[6], __mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6]);
//}
//__global__ void simple_test_school_inplace_8x8_low_v2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 1;
//	__school32_inplace_8x8_low(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7], __mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6],__mu[7]);
//}
//
//
//__global__ void simple_test_reduction_inplace2(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
////	__reduce32_inplace4(out[0],out[1],out[2],out[3],
////			__mu[0],__mu[1],
////			__p[0],__p[1]);
//}
//__global__ void simple_test_reduction_inplace3(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
////	__reduce32_inplace6(out[0],out[1],out[2],out[3],out[4],out[5],
////			__mu[0],__mu[1],__mu[2],
////			__p[0],__p[1],__p[2]);
//}
//__global__ void simple_test_reduction_inplace4(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
////	__reduce32_inplace8(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],
////			__mu[0],__mu[1],__mu[2],__mu[3],
////			__p[0],__p[1],__p[2],__p[3]);
//}
//__global__ void simple_test_reduction_inplace5(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
////	__reduce32_inplace10(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],
////			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4], 
////			__p[0],__p[1],__p[2],__p[3],__p[4]);
//}
//__global__ void simple_test_reduction_inplace6(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
//	__reduce32_inplace6x6(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5], 
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5]);
//}
//__global__ void simple_test_reduction_inplace7(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
//	__reduce32_inplace7x7(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6], 
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5],__p[6]);
//}
//__global__ void simple_test_reduction_inplace8(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	out[0] = 0; 
//	__reduce32_inplace16(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13],out[14],out[15], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6],__mu[7],
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5],__p[6],__p[7]);
//}
//
//__global__ void simple_test_reduction4(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
//	__reduce32_8(out[0],out[1],out[2],out[3],
//			out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],
//			__mu[0],__mu[1],__mu[2],__mu[3], 
//			__p[0],__p[1],__p[2],__p[3]);
//}
//
//__global__ void simple_test_reduction6(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
//	__reduce32_12(out[0],out[1],out[2],out[3],out[4],out[5],
//			out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5], 
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5]);
//}
//
//__global__ void simple_test_reduction7(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }
//	
//	out[0] = 0;
//	__reduce32_14(out[0],out[1],out[2],out[3],out[4],out[5],out[6],
//			out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6], 
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5],__p[6]);
//}
//
//__global__ void simple_test_reduction8(uint32_t *out, uint32_t p1){
//	for(uint32_t i = 0; i < 16; i++){ out[i] = 0; }	
//	out[0] = 0; 
//	__reduce32_16(out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],
//			out[0],out[1],out[2],out[3],out[4],out[5],out[6],out[7],out[8],out[9],out[10],out[11],out[12],out[13],out[14],out[15], 
//			__mu[0],__mu[1],__mu[2],__mu[3],__mu[4],__mu[5],__mu[6],__mu[7], 
//			__p[0],__p[1],__p[2],__p[3],__p[4],__p[5],__p[6],__p[7]);
//}
//
//
//#ifndef NOMAIN
//int main() { 
//	constexpr uint32_t size = 16;
//	uint32_t out[size]  = {0};
//	uint32_t out2[size] = {0};
//	uint32_t out3[size] = {0};
//	uint32_t out4[size] = {0};
//	uint32_t out5[size] = {0};
//	uint32_t out6[size] = {0};
//	uint32_t out7[size] = {0};
//	uint32_t out8[size] = {0};
//	uint32_t out9[size] = {0};
//
//	uint32_t out10[size] = {0};
//	uint32_t out11[size] = {0};
//	uint32_t out12[size] = {0};
//	uint32_t out13[size] = {0};
//	uint32_t out14[size] = {0};
//	
//	uint32_t out15[size] = {0};
//	uint32_t out16[size] = {0};
//	uint32_t out17[size] = {0};
//	uint32_t out18[size] = {0};
//	uint32_t out19[size] = {0};
//
//	uint32_t out20[size] = {0};
//	uint32_t out21[size] = {0};
//	uint32_t out22[size] = {0};
//	
//	uint32_t out23[size] = {0};
//	uint32_t out24[size] = {0};
//
//	uint32_t out25[size] = {0};
//	
//	uint32_t out26[size] = {0};
//	uint32_t out27[size] = {0};
//
//	uint32_t out28[size] = {0};
//	uint32_t out29[size] = {0};
//	uint32_t out30[size] = {0};
//	
//	uint32_t out31[size] = {0};
//	uint32_t out32[size] = {0};
//	uint32_t out33[size] = {0};
//	uint32_t out34[size] = {0};
//	uint32_t out35[size] = {0};
//	uint32_t *d_out;
//
//	hipError_t errcode;
//	if (hipSuccess != (errcode = hipMalloc((void**)&d_out, sizeof(uint32_t) * size))) {
//		printf ("hipMalloc failed %s:%d\n", __FILE__, __LINE__);
//		printf ("   %s\n", hipGetErrorString(errcode));
//		return;
//	}
//
//	const uint32_t p1 = 1u << 31u; 
//	// simple_test1<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	//      
//	// simple_test2<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out2, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test2_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out26, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	// simple_test3<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out3, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	// 
//	// simple_test4<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out4, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	//     	
//	// simple_test5<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out5, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	//	
//	// simple_test6<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out6, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	//	
//	// simple_test7<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out7, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test8<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out25, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	  
//	// simple_test44<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out8, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	//     	
//	// simple_test33<<<1,1>>>(d_out, p1);
//	// hipDeviceSynchronize();
//	// hipMemcpy(out9, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test6444<<<1,1>>>((uint64_t *)d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out23, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	    	
//	simple_test6433<<<1,1>>>((uint64_t *)d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out24, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//
//	simple_test_school_inplace_3x3_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out28, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 3; i > 0; i--){
//		printf("%x ", out28[i-1]);
//	}
//	printf(" school_inplace_3x3_low\n");	
//
//	simple_test_school_inplace_4x4_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out29, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 4; i > 0; i--){
//		printf("%x ", out29[i-1]);
//	}
//	printf(" school_inplace_4x4_low\n");	
//
//	simple_test_school_inplace_5x5_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out30, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 5; i > 0; i--){
//		printf("%x ", out30[i-1]);
//	}
//	printf(" school_inplace_5x5_low\n");
//
//
//	simple_test_school_inplace_6x6_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out15, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 6; i > 0; i--){
//		printf("%x ", out15[i-1]);
//	}
//	printf(" school_inplace_6x6_low\n");
//
//	for(uint32_t i = 6; i > 0; i--){
//		printf("%x ", __cpu_mu[i-1]);
//	}
//	printf(" mu\n");
//
//	simple_test_school_inplace_7x7_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out16, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 7; i > 0; i--){
//		printf("%x ", out16[i-1]);
//	}
//	printf(" school_inplace_7x7_low\n");
//
//	for(uint32_t i = 7; i > 0; i--){
//		printf("%x ", __cpu_mu[i-1]);
//	}
//	printf(" mu\n");
//	simple_test_school_inplace_8x8_low_v2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out17, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	for(uint32_t i = 8; i > 0; i--){
//		printf("%x ", out17[i-1]);
//	}
//	printf(" school_inplace_8x8_low\n");
//
//	for(uint32_t i = 8; i > 0; i--){
//		printf("%x ", __cpu_mu[i-1]);
//	}
//	printf(" mu\n");
//
//	simple_test_school5x5<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out31, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_school6x6<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out10, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_school7x7<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out18, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
////	simple_test_fp_school6x6<<<1,1>>>(d_out, p1);
////	hipDeviceSynchronize();
////	hipMemcpy(out11, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_school8x8<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out12, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//
//	simple_test_reduction_inplace2<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out32, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_reduction_inplace3<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out33, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_reduction_inplace4<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out34, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_reduction_inplace5<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out35, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_reduction_inplace6<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out13, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_reduction_inplace7<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out19, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_reduction_inplace8<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out14, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_reduction4<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out27, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	simple_test_reduction6<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out20, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_reduction7<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out21, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//	
//	simple_test_reduction8<<<1,1>>>(d_out, p1);
//	hipDeviceSynchronize();
//	hipMemcpy(out22, d_out, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out31[i-1]);
//	}
//	printf(" school 5x5\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out10[i-1]);
//	}
//	printf(" school 6x6\n");
//
////	for(uint32_t i = size; i > 0; i--){
////		printf("%u ", out11[i-1]);
////	}
////	printf(" fp school 6x6\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out18[i-1]);
//	}
//	printf(" school 7x7\n");
//	
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out12[i-1]);
//	}
//	printf(" school 8x8\n");
//	
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out32[i-1]);
//	}
//	printf(" reduce inplace 2\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out33[i-1]);
//	}
//	printf(" reduce inplace 3\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out34[i-1]);
//	}
//	printf(" reduce inplace 4\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out35[i-1]);
//	}
//	printf(" reduce inplace 5\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out13[i-1]);
//	}
//	printf(" reduce inplace 6\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out19[i-1]);
//	}
//	printf(" reduce inplace 7\n");
//	
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out14[i-1]);
//	}
//	printf(" reduce inplace 8\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out27[i-1]);
//	}
//	printf(" reduce 4\n");
//	
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out20[i-1]);
//	}
//	printf(" reduce 6\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out21[i-1]);
//	}
//	printf(" reduce 7\n");
//	
//	for(uint32_t i = size; i > 0; i--){
//		printf("%x ", out22[i-1]);
//	}
//	printf(" reduce 8\n");
//
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out23[i-1]);
//	}
//	printf(" 4x4 64\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out24[i-1]);
//	}
//	printf(" 3x3 64\n");
//
//
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out26[i-1]);
//	}
//	printf(" 1x2 v2\n");
//
//
//
//
//	return 0;
//	
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out[i-1]);
//	}
//	printf(" 2x2\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out2[i-1]);
//	}
//	printf(" 1x2\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out3[i-1]);
//	}
//	printf(" 1x3\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out4[i-1]);
//	}
//	printf(" 1x4\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out5[i-1]);
//	}
//	printf(" 1x5\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out6[i-1]);
//	}
//	printf(" 1x6\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out7[i-1]);
//	}
//	printf(" 1x7\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out25[i-1]);
//	}
//	printf(" 1x25\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out8[i-1]);
//	}
//	printf(" 4x4\n");
//
//	for(uint32_t i = size; i > 0; i--){
//		printf("%u ", out9[i-1]);
//	}
//	printf(" 3x3\n");
//
//	hipFree (d_out);
//	return 0;
//}
//#endif
