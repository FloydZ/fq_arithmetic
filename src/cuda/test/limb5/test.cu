#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


#include "test_vecs.cuh"
#include "../../src/arith.cuh"


static inline int compare(const limb_t *a, 
		const limb_t *b, 
		const uint32_t len=LIMBS) {     
	for (uint32_t i = 0; i < len; i++) {
		if (a[i] != b[i]) {
			return 1; 
		}	
	}

	return 0;
}

__global__ void add_fp(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[LIMBS] = {0};

    __fp_add160_32(c[0], c[1], c[2], c[3], c[4],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  b[i][0], b[i][1], b[i][2], b[i][3], b[i][4],
            	  __p[0], __p[1], __p[2], __p[3], __p[4]);

    for (uint32_t j = 0; j < LIMBS; j++) {
		out[i*LIMBS + j] = c[j];
	}
}

__global__ void sub_fp(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[LIMBS] = {0};

    __fp_sub160_32(c[0], c[1], c[2], c[3], c[4],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  b[i][0], b[i][1], b[i][2], b[i][3], b[i][4],
            	  __p[0], __p[1], __p[2], __p[3], __p[4]);

    for (uint32_t j = 0; j < LIMBS; j++) {
		out[i*LIMBS + j] = c[j];
	}
}

__global__ void schoolbookmul(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[2*LIMBS] = {0};

    __school32_5x5(c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  b[i][0], b[i][1], b[i][2], b[i][3], b[i][4]);

    for (uint32_t j = 0; j < 2*LIMBS; j++) {
		out[2*i*LIMBS + j] = c[j];
	}
}

__global__ void mul(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[2*LIMBS] = {0};

    __mul160_32(c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  b[i][0], b[i][1], b[i][2], b[i][3], b[i][4]);

    for (uint32_t j = 0; j < 2*LIMBS; j++) {
		out[2*i*LIMBS + j] = c[j];
	}
}

__global__ void schoolbookmullow(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[LIMBS] = {0};

    __school32_5x5_low(c[0], c[1], c[2], c[3], c[4],
				  	   a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  	   b[i][0], b[i][1], b[i][2], b[i][3], b[i][4]);

    for (uint32_t j = 0; j < LIMBS; j++) {
		out[i*LIMBS + j] = c[j];
	}
}

__global__ void mul_fp(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[LIMBS] = {0};

    __fp_mul5x5_32(c[0], c[1], c[2], c[3], c[4],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4],
				  b[i][0], b[i][1], b[i][2], b[i][3], b[i][4],
				  __mu[0], __mu[1], __mu[2], __mu[3], __mu[4],
            	  __p[0], __p[1], __p[2], __p[3], __p[4]);

    for (uint32_t j = 0; j < LIMBS; j++) {
		out[i*LIMBS + j] = c[j];
	}
}


__global__ void sqr_fp(limb_t *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    limb_t c[2*LIMBS] = {0};
   
	__sqr160_32(c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9],
				  a[i][0], a[i][1], a[i][2], a[i][3], a[i][4])
    
	for (uint32_t j = 0; j < 2*LIMBS; j++) {
		out[2*i*LIMBS + j] = c[j];
	}

}


void test_add_fp() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing Addition in Fp: ");
    hipMallocManaged(&c, TESTS * LIMBS * sizeof(limb_t));

    add_fp<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[i*LIMBS], add_fp_ok[i])) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[i*LIMBS + 4], c[i*LIMBS + 3], c[i*LIMBS + 2], c[i*LIMBS + 1], c[i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", add_fp_ok[i][4], add_fp_ok[i][3], add_fp_ok[i][2], add_fp_ok[i][1], add_fp_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_sub_fp() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing Subtraction in Fp: ");
    hipMallocManaged(&c, TESTS * LIMBS * sizeof(limb_t));

    sub_fp<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[i*LIMBS], sub_fp_ok[i])) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[i*LIMBS + 4], c[i*LIMBS + 3], c[i*LIMBS + 2], c[i*LIMBS + 1], c[i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", sub_fp_ok[i][4], sub_fp_ok[i][3], sub_fp_ok[i][2], sub_fp_ok[i][1], sub_fp_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_mul() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing Multiplication: ");
    hipMallocManaged(&c, 2 * TESTS * LIMBS * sizeof(limb_t));

    mul<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[2*i*LIMBS], mul_ok[i], 2*LIMBS)) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[2*i*LIMBS + 9], c[2*i*LIMBS + 8], c[2*i*LIMBS + 7], c[2*i*LIMBS + 6], c[2*i*LIMBS + 5], c[2*i*LIMBS + 4], c[2*i*LIMBS + 3], c[2*i*LIMBS + 2], c[2*i*LIMBS + 1], c[2*i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX\n", mul_ok[i][9], mul_ok[i][8], mul_ok[i][7], mul_ok[i][6], mul_ok[i][5], mul_ok[i][4], mul_ok[i][3], mul_ok[i][2], mul_ok[i][1], mul_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_schoolbookmullow() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing SchoolBook Multiplication Low: ");
    hipMallocManaged(&c, TESTS * LIMBS * sizeof(limb_t));

    schoolbookmullow<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[i*LIMBS], mul_ok[i])) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[i*LIMBS + 4],  c[i*LIMBS + 3], c[i*LIMBS + 2], c[i*LIMBS + 1], c[i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", mul_ok[i][4], mul_ok[i][3], mul_ok[i][2], mul_ok[i][1], mul_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_schoolbookmul() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing SchoolBook Multiplication: ");
    hipMallocManaged(&c, 2 * TESTS * LIMBS * sizeof(limb_t));

    schoolbookmul<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[2*i*LIMBS], mul_ok[i], 2*LIMBS)) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[2*i*LIMBS + 9], c[2*i*LIMBS + 8], c[2*i*LIMBS + 7], c[2*i*LIMBS + 6], c[2*i*LIMBS + 5], c[2*i*LIMBS + 4], c[2*i*LIMBS + 3], c[2*i*LIMBS + 2], c[2*i*LIMBS + 1], c[2*i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX %.8lX\n", mul_ok[i][9], mul_ok[i][8], mul_ok[i][7], mul_ok[i][6], mul_ok[i][5], mul_ok[i][4], mul_ok[i][3], mul_ok[i][2], mul_ok[i][1], mul_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_mul_fp() {
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing Multiplication in Fp: ");
    hipMallocManaged(&c, TESTS * LIMBS * sizeof(limb_t));

    mul_fp<<<blocks, threads>>>(c);
    hipDeviceSynchronize();

    for (uint32_t i = 0; i < TESTS; i++) {
        if (compare(&c[i*LIMBS], mul_fp_ok[i])) {
            printf("(%i) FAIL\n", i);
            printf("(%d) 0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", i, c[i*LIMBS + 4], c[i*LIMBS + 3], c[i*LIMBS + 2], c[i*LIMBS + 1], c[i*LIMBS + 0]);
            printf("ok  0x%.8lX %.8lX %.8lX %.8lX %.8lX\n", mul_fp_ok[i][4], mul_fp_ok[i][3], mul_fp_ok[i][2], mul_fp_ok[i][1], mul_fp_ok[i][0]);
            return;
        }
    }
    printf("OK\n");

    hipFree(c);
}

void test_sqr_fp() {
    int i;
    uint32_t blocks = 1;
    uint32_t threads = TESTS;
    limb_t *c;

    if (threads > 1024) {
        while (threads <= 1024) {
            threads >>= 1;
            blocks += 1;
        }
    }

    printf("Testing Square in Fp: ");
    hipMallocManaged(&c, TESTS * sizeof(limb_t));

    sqr_fp<<<blocks, threads>>>(c);

    hipDeviceSynchronize();

    for (i = 0; i < TESTS; i++) {
        if (compare(&c[i], sqr_fp_ok[i])) {
            printf("(%i) FAIL\n", i);
            /*
            printf("(%d) 0x%.16lX%.16lX%.16lX\n", i, c[2],c[1],c[0]);
            printf("ok 0x%.16lX%.16lX%.16lX\n",
                    mul_fp_ok[i][2],mul_fp_ok[i][1],mul_fp_ok[i][0]);
            */
            return;
        }
    }

    printf("OK\n");
    hipFree(c);
}

void test_arith() {
    test_add_fp();
    test_sub_fp();
	test_mul();
	test_schoolbookmullow();
	test_schoolbookmul();
    test_mul_fp();
    //test_sqr_fp();
}


int main() {
	test_arith();
}
